// compile with nime nvcc -o sign sign.cu  -rdc=false -Xptxas -v  -O0  -lineinfo --ptxas-options=-O0
// /usr/bin/c++ -DSODIUM_STATIC -I/home/urllc2/bls-signatures/src -I/home/urllc2/bls-signatures/build/_deps/relic-src/include -I/home/urllc2/bls-signatures/build/_deps/relic-build/include -I/home/urllc2/bls-signatures/build/_deps/sodium-src/libsodium/src/libsodium/include -O3 -DNDEBUG -fPIE -std=gnu++17 -MD -MT main.cpp.o -MF main.cpp.o.d -o main.cpp.o -c main.cpp; /usr/bin/c++ -O3 -DNDEBUG main.cpp.o -o runmain  /home/urllc2/bls-signatures/build/src/libbls.a /home/urllc2/bls-signatures/build/_deps/relic-build/lib/librelic_s.a /usr/lib/x86_64-linux-gnu/libgmp.so -lrt -lpthread -lm /home/urllc2/bls-signatures/build/_deps/sodium-build/libsodium.a
//  sudo apt-get purge nvidia*
//  sudo apt-get autoremove
//  sudo reboot
//  lsmod | grep nvidia.drm
//  sudo sh cuda_12.0.0_525.60.13_linux.run
//  sudo /usr/local/NVIDIA-Nsight-Compute-2022.4/ncu --call-stack -f --set detailed -k saxpy -o res ./sign --metrics gpu__time_duration.sum


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>
#include <time.h>

#include <inttypes.h>
#include <stdint.h>
#include<string.h>
#include <stdarg.h>
#include <ctype.h>


#define NBLOCKS 1
#define NTHREADS 1

#define INLINE 0
/** Prime field size in bits. */
#define FP_PRIME 381
#define RLC_BN_SIZE 8
#define RLC_DIG 64
#define fp_null(A)                      /* empty */
#define fp_new(A)                       /* empty */
#define bn_null(A)                      /* empty */
#define BASIC    1
#define FP_RDC   BASIC
#define PROJC    2
/** Use -1 as quadratic non-residue. */
#define FP_QNRES


/** Size of word in this architecture. */
#define WSIZE    64

//#define RLC_FP_DIGS             (FP_PRIME/WSIZE + 1)
/**
 * Computes the ceiling function of an integer division.
 *
 * @param[in] A                 - the dividend.
 * @param[in] B                 - the divisor.
 */
#define RLC_CEIL(A, B)                  (((A) - 1) / (B) + 1)
/**
 * Returns the given character in upper case.
 *
 * @param[in] C                 - the character. 
 */
#define RLC_UPP(C)                              ((C) - 0x20 * (((C) >= 'a') && ((C) <= 'z')))
/**
 * Precision in bits of a prime field element.
 */
#define RLC_FP_BITS     ((int)FP_PRIME)
/**
 * Size in digits of a block sufficient to store a prime field element.
 */
#define RLC_FP_DIGS     ((int)RLC_CEIL(RLC_FP_BITS, RLC_DIG))
/**
 * Size in bytes of a block sufficient to store a binary field element.
 */
#define RLC_FP_BYTES    ((int)RLC_CEIL(RLC_FP_BITS, 8))
/**
 * Maximum number of coefficients of an isogeny map polynomial.
 * 4 is sufficient for a degree-3 isogeny polynomial.
 */
#define RLC_EPX_CTMAP_MAX       4
#define BASIC    1
/** @{ */
#define B12_P381_A0             "0"
#define B12_P381_A1             "0"
#define B12_P381_B0             "4"
#define B12_P381_B1             "4"
#define B12_P381_X0             "024AA2B2F08F0A91260805272DC51051C6E47AD4FA403B02B4510B647AE3D1770BAC0326A805BBEFD48056C8C121BDB8"
#define B12_P381_X1             "13E02B6052719F607DACD3A088274F65596BD0D09920B61AB5DA61BBDC7F5049334CF11213945D57E5AC7D055D042B7E"
#define B12_P381_Y0             "0CE5D527727D6E118CC9CDC6DA2E351AADFD9BAA8CBDD3A76D429A695160D12C923AC9CC3BACA289E193548608B82801"
#define B12_P381_Y1             "0606C4A02EA734CC32ACD2B02BC28B99CB3E287E85A763AF267492AB572E99AB3F370D275CEC1DA1AAA9075FF05F79BE"
#define B12_P381_R              "73EDA753299D7D483339D80809A1D80553BDA402FFFE5BFEFFFFFFFF00000001"
#define B12_P381_H              "5D543A95414E7F1091D50792876A202CD91DE4547085ABAA68A205B2E5A7DDFA628F1CB4D9E82EF21537E293A6691AE1616EC6E786F0C70CF1C38E31C7238E5"
#define B12_P381_ISO_A0 "0"
#define B12_P381_ISO_A1 "F0"
#define B12_P381_ISO_B0 "3F4"
#define B12_P381_ISO_B1 "3F4"
#define B12_P381_ISO_XN "5c759507e8e333ebb5b7a9a47d7ed8532c52d39fd3a042a88b58423c50ae15d5c2638e343d9c71c6238aaaaaaaa97d6,5c759507e8e333ebb5b7a9a47d7ed8532c52d39fd3a042a88b58423c50ae15d5c2638e343d9c71c6238aaaaaaaa97d6;0,11560bf17baa99bc32126fced787c88f984f87adf7ae0c7f9a208c6b4f20a4181472aaa9cb8d555526a9ffffffffc71a;11560bf17baa99bc32126fced787c88f984f87adf7ae0c7f9a208c6b4f20a4181472aaa9cb8d555526a9ffffffffc71e,8ab05f8bdd54cde190937e76bc3e447cc27c3d6fbd7063fcd104635a790520c0a395554e5c6aaaa9354ffffffffe38d;171d6541fa38ccfaed6dea691f5fb614cb14b4e7f4e810aa22d6108f142b85757098e38d0f671c7188e2aaaaaaaa5ed1,0"
#define B12_P381_ISO_XD "0,1a0111ea397fe69a4b1ba7b6434bacd764774b84f38512bf6730d2a0f6b0f6241eabfffeb153ffffb9feffffffffaa63;c,1a0111ea397fe69a4b1ba7b6434bacd764774b84f38512bf6730d2a0f6b0f6241eabfffeb153ffffb9feffffffffaa9f;1,0"
#define B12_P381_ISO_YN "1530477c7ab4113b59a4c18b076d11930f7da5d4a07f649bf54439d87d27e500fc8c25ebf8c92f6812cfc71c71c6d706,1530477c7ab4113b59a4c18b076d11930f7da5d4a07f649bf54439d87d27e500fc8c25ebf8c92f6812cfc71c71c6d706;0,5c759507e8e333ebb5b7a9a47d7ed8532c52d39fd3a042a88b58423c50ae15d5c2638e343d9c71c6238aaaaaaaa97be;11560bf17baa99bc32126fced787c88f984f87adf7ae0c7f9a208c6b4f20a4181472aaa9cb8d555526a9ffffffffc71c,8ab05f8bdd54cde190937e76bc3e447cc27c3d6fbd7063fcd104635a790520c0a395554e5c6aaaa9354ffffffffe38f;124c9ad43b6cf79bfbf7043de3811ad0761b0f37a1e26286b0e977c69aa274524e79097a56dc4bd9e1b371c71c718b10,0"
#define B12_P381_ISO_YD "1a0111ea397fe69a4b1ba7b6434bacd764774b84f38512bf6730d2a0f6b0f6241eabfffeb153ffffb9feffffffffa8fb,1a0111ea397fe69a4b1ba7b6434bacd764774b84f38512bf6730d2a0f6b0f6241eabfffeb153ffffb9feffffffffa8fb;0,1a0111ea397fe69a4b1ba7b6434bacd764774b84f38512bf6730d2a0f6b0f6241eabfffeb153ffffb9feffffffffa9d3;12,1a0111ea397fe69a4b1ba7b6434bacd764774b84f38512bf6730d2a0f6b0f6241eabfffeb153ffffb9feffffffffaa99;1,0"
#define B12_P381_MAPU0 "-2"
#define B12_P381_MAPU1 "-1"
__device__
#if INLINE == 0
__noinline__
#endif
void print_multiple_precision(const uint64_t *number, int size){
// This function prints a multiple precision integer stored as uint64_t
  for(int i=0; i < size; i++){
   printf("%" PRIu64 "\n", number[i]);
  }
}
__device__
#if INLINE == 0
__noinline__
#endif
void print_line(){
// This function prints a line
  printf("\n-----------------\n");
}
/** @} */
__device__ 
__noinline__
inline static unsigned int lzcnt64_generic(unsigned long long x)
{
    unsigned int n;
    static unsigned int clz_table_4[] = {
        0,
        4,
        3, 3,
        2, 2, 2, 2,
        1, 1, 1, 1, 1, 1, 1, 1
    };

    if (x == 0) {
        return sizeof(x)*8;
    }

    n = clz_table_4[x >> (sizeof(x)*8 - 4)];
    if (n == 0) {
        if ((x & ((unsigned long long)0xFFFFFFFF << 32)) == 0) { n  = 32; x <<= 32; }
        if ((x & ((unsigned long long)0xFFFF0000 << 32)) == 0) { n += 16; x <<= 16; }
        if ((x & ((unsigned long long)0xFF000000 << 32)) == 0) { n += 8;  x <<= 8;  }
        if ((x & ((unsigned long long)0xF0000000 << 32)) == 0) { n += 4;  x <<= 4;  }
        n += clz_table_4[x >> (sizeof(x)*8 - 4)];
    }

    return n - 1;
}
__device__ 
__noinline__
unsigned int lzcnt32_generic(unsigned int x)
{
    unsigned int n; 
    static unsigned int clz_table_4[] = {
        0,
        4, 
        3, 3, 
        2, 2, 2, 2, 
        1, 1, 1, 1, 1, 1, 1, 1
    };

    if (x == 0) {
        return sizeof(x)*8;
    }

    n = clz_table_4[x >> (sizeof(x)*8 - 4)];
    if (n == 0) {
        if ((x & 0xFFFF0000) == 0) { n  = 16; x <<= 16; }
        if ((x & 0xFF000000) == 0) { n += 8;  x <<= 8;  }
        if ((x & 0xF0000000) == 0) { n += 4;  x <<= 4;  }
        n += clz_table_4[x >> (sizeof(x)*8 - 4)];
    }
    return n - 1;
}
/**
 * Returns the minimum between two numbers.
 *
 * @param[in] A         - the first number.
 * @param[in] B         - the second number.
 */
#define RLC_MIN(A, B)                   ((A) < (B) ? (A) : (B))
/**
 * Negative sign of a multiple precision integer.
 */
#define RLC_NEG         1
/**
 * Returns the maximum between two numbers.
 *
 * @param[in] A         - the first number.
 * @param[in] B         - the second number.
 */
#define RLC_MAX(A, B)                   ((A) > (B) ? (A) : (B))
#define FB_POLYN 283
#define RLC_DV_MAX              (FB_POLYN)
/** Irreducible polynomial size in bits. */
//#define RLC_DV_DIGS             (RLC_MAX(RLC_CEIL(RLC_DV_MAX, RLC_DIG), RLC_BN_SIZE))
#define RLC_DV_DIGS             (2 * RLC_FP_DIGS + 1)
/**
 * List of possible errors generated by the library.
 */             
enum errors {   
        /** Constant to indicate the first an error already catched. */
        ERR_CAUGHT = 1,
        /** Occurs when memory-allocating functions fail. */
        ERR_NO_MEMORY,
        /** Occcurs when the library precision is not sufficient. */
        ERR_NO_PRECI,
        /** Occurs when a file is not found. */
        ERR_NO_FILE,
        /** Occurs when the specified number of bytes cannot be read from source. */
        ERR_NO_READ,
        /** Occurs when an invalid value is passed as input. */
        ERR_NO_VALID,
        /** Occurs when a buffer capacity is insufficient. */
        ERR_NO_BUFFER,  
        /** Occurs when there is not a supported field in the security level. */
        ERR_NO_FIELD,
        /** Occurs when there is not a supported curve in the security level. */
        ERR_NO_CURVE,
        /** Occurs when the library configuration is incorrect. */
        ERR_NO_CONFIG,
        /** Occurs when the PRNG is stuck at one value. */
        ERR_NO_RAND,
        /** Constant to indicate the number of errors. */
        ERR_MAX
};

/**     
 * Indicates that the function executed correctly.
 */     
#define RLC_OK                  0

/**     
 * Indicates that an error occurred during the function execution.
 */     
#define RLC_ERR                 1 
                
/**     
 * Indicates that a comparison returned that the first argument was lesser than
 * the second argument.
 */
#define RLC_LT                  -1
        
/**
 * Indicates that a comparison returned that the first argument was equal to
 * the second argument.
 */     
#define RLC_EQ                  0
        
/**
 * Indicates that a comparison returned that the first argument was greater than
 * the second argument.
 */     
#define RLC_GT                  1

/**
 * Indicates that two incomparable elements are not equal.
 */
#define RLC_NE                  2

/**
 * Optimization identifer for the case where a coefficient is 0.
 */
#define RLC_ZERO                0

/** 
 * Optimization identifer for the case where a coefficient is 1.
 */ 
#define RLC_ONE                 1
    
/** 
 * Optimization identifer for the case where a coefficient is 2.
 */
#define RLC_TWO                 2
typedef uint64_t dig_t;
typedef __uint128_t dbl_t;
typedef dig_t *fp_t;
typedef dig_t *dv_t;
#define RLC_PAD(A)              (0)
/**
 * Represents a prime field element with automatic memory allocation.
 */
typedef dig_t fp_st[RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)];
/**
 * Maximum number of coefficients of an isogeny map polynomial.
 * RLC_TERMS of value 16 is sufficient for a degree-11 isogeny polynomial.
 */
#define RLC_EP_CTMAP_MAX                16
/**
 * Represents a quadratic extension prime field element.
 *
 * This extension is constructed with the basis {1, i}, where i is an adjoined
 * square root in the prime field.
 */
typedef fp_t fp2_t[2];
/**
 * Coefficients of an isogeny map for a curve over a quadratic extension.
 */
typedef struct {
        /** The a-coefficient of the isogenous curve used for SSWU mapping. */
        fp2_t a;
        /** The b-coefficient of the isogenous curve used for SSWU mapping. */
        fp2_t b;
        /** Degree of x numerator */
        int deg_xn;
        /** Degree of x denominator */
        int deg_xd;
        /** Degree of y numerator */
        int deg_yn;
        /** Degree of y denominator */
        int deg_yd;
        /** x numerator coefficients */
        fp2_t xn[RLC_EPX_CTMAP_MAX];
        /** x denominator coefficients */
        fp2_t xd[RLC_EPX_CTMAP_MAX];
        /** y numerator coefficients */
        fp2_t yn[RLC_EPX_CTMAP_MAX];
        /** y denominator coefficients */
        fp2_t yd[RLC_EPX_CTMAP_MAX];
} iso2_st;
/**
 * Pointer to isogeny map coefficients.
 */
typedef iso2_st *iso2_t;
/**
 * Represents an elliptic curve point over a quadratic extension over a prime
 * field.
 */
typedef struct {
        /** The first coordinate. */
        fp2_t x;
        /** The second coordinate. */
        fp2_t y;
        /** The third coordinate (projective representation). */
        fp2_t z;
        /** Flag to indicate the coordinate system of this point. */
        int coord;
} ep2_st;
typedef ep2_st *ep2_t;
__device__
#if INLINE == 0
__noinline__
#endif
int util_bits_dig(dig_t a) {
    return RLC_DIG - lzcnt64_generic(a);
}
/**
 * Represents a multiple precision integer.
 *
 * The field dp points to a vector of digits. These digits are organized
 * in little-endian format, that is, the least significant digits are
 * stored in the first positions of the vector.
 */
typedef struct {
        /** The number of digits allocated to this multiple precision integer. */
        int alloc;
        /** The number of digits actually used. */
        int used;
        /** The sign of this multiple precision integer. */
        int sign;
        dig_t *dp;
} bn_st;
typedef bn_st *bn_t;
/**
 * Positive sign of a multiple precision integer.
 */
#define RLC_POS         0

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code,
                      const char *file,
                      int line,
                      bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n",
            hipGetErrorString(code), file, line);
    exit(code);
  }
}

__device__ __constant__ dig_t fp_prime[10];
//////////////////////////////////////////////////////////////
// CUDA shared memory declarations and static allocation    //
  __shared__ dig_t* shared_u;
  __shared__ dig_t shared_m[10];
  __shared__ dig_t shared_prime[10];
  __shared__ bn_t shared_prime_bn;
  __shared__ dig_t shared_one[10];
  __shared__ dig_t shared_conv[10];
  __shared__ fp_t shared_a[10];
  __shared__ fp_t shared_b[10];
  __shared__ fp2_t shared_map_u;
  __shared__ fp2_t shared_c[4];
  __shared__ iso2_t shared_coeffs;
//////////////////////////////////////////////////////////////
/**
 * Multiplies two digits to give a double precision result.
 *
 * @param[out] H                - the higher half of the result.
 * @param[out] L                - the lower half of the result.
 * @param[in] A                 - the first digit to multiply.
 * @param[in] B                 - the second digit to multiply.
 */
#define RLC_MUL_DIG(H, L, A, B)                     \
        H = ((dbl_t)(A) * (dbl_t)(B)) >> RLC_DIG;   \
        L = (A) * (B);                              \
/**
 * Accumulates a single precision digit in a triple register variable.
 *
 * @param[in,out] T                     - the temporary variable.
 * @param[in,out] R2            - most significant word of the triple register.
 * @param[in,out] R1            - middle word of the triple register.
 * @param[in,out] R0            - lowest significant word of the triple register.
 * @param[in] A                         - the first digit to accumulate.
 */
#define RLC_COMBA_ADD(T, R2, R1, R0, A)  \
        (T) = (R1);                      \
        (R0) += (A);                     \
        (R1) += (R0) < (A);              \
        (R2) += (R1) < (T);              \


/**     
 * Accumulates a double precision digit in a triple register variable.
 *
 * @param[in,out] R2            - most significant word of the triple register.
 * @param[in,out] R1            - middle word of the triple register.
 * @param[in,out] R0            - lowest significant word of the triple register.
 * @param[in] A                         - the first digit to multiply.
 * @param[in] B                         - the second digit to multiply.
 */             
#define RLC_COMBA_STEP_MUL(R2, R1, R0, A, B)  \
        dig_t _r, _r0, _r1;                   \
        RLC_MUL_DIG(_r1, _r0, A, B);          \
        RLC_COMBA_ADD(_r, R2, R1, R0, _r0);   \
        (R1) += _r1;                          \
        (R2) += (R1) < _r1;                   \


/**
 * Returns a bit mask to isolate the lowest part of a digit.
 *
 * @param[in] B                 - the number of bits to isolate.
 */
#define RLC_MASK(B)                                                                                                                     \
        ((-(dig_t)((B) >= WSIZE)) | (((dig_t)1 << ((B) % WSIZE)) - 1))
#define RLC_TRY                                 if (1)
#define RLC_CATCH_ANY                   if (0)
#define RLC_THROW                               printf("error thrown in operation...\n")
#define RLC_FINALLY                             if (1)
#define RLC_DIG_LOG             6
/**
 * Splits a bit count in a digit count and an updated bit count.
 *              
 * @param[out] B                - the resulting bit count.
 * @param[out] D                - the resulting digit count.
 * @param[out] V                - the bit count.
 */
#define RLC_RIP(B, D, V)                                                                                                        \
        D = (V) >> (RLC_DIG_LOG); B = (V) - ((D) * (1 << RLC_DIG_LOG));
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__
#if INLINE == 0
__noinline__
#endif
void dv_copy(dig_t *c, const dig_t *a, int digits) {
//        printf("now copying data: %d \n ", digits);
        memcpy(c, a, digits * sizeof(dig_t));
//        printf("data copied data: %d \n", digits);
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t* cuda_realloc(int oldsize, int newsize, dig_t* old)
{
    dig_t* newT = (dig_t*) malloc (newsize*sizeof(dig_t));
    for(int i=0; i<oldsize; i++)
    {
        newT[i] = old[i];
    }

    free(old);
    return newT;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_grow(bn_t a, int digits) {
//#if ALLOC == DYNAMIC
        dig_t *t;

        if (a->alloc < digits) {
                /* At least add RLC_BN_SIZE more digits. */
                digits += (RLC_BN_SIZE * 2) - (digits % RLC_BN_SIZE);
                t = (dig_t *)cuda_realloc(a->alloc, (RLC_DIG / 8) * digits, a->dp);
                if (t == NULL) {
                        printf("no more memory in bn_grow...\n ");
                        return;
                }
                a->dp = t;
                /* Set the newly allocated digits to zero. */
                a->alloc = digits;
        }
//#elif ALLOC == AUTO
//        if (digits > RLC_BN_SIZE) {
//                printf("bn_grow error, RLC_BN_SIZE: %d digits: %d .... \n", RLC_BN_SIZE, digits);
//                return;
//        }
//        (void)a;
//#endif
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_copy(bn_t c, const bn_t a) {
        if (c->dp == a->dp) {
                return;
        }
        bn_grow(c, a->used);
        dv_copy(c->dp, a->dp, a->used);
        c->used = a->used;      
        c->sign = a->sign;
}
__device__
#if INLINE == 0
__noinline__
#endif
void dv_lshd(dig_t *c, const dig_t *a, int size, int digits) {
        dig_t *top;
        const dig_t *bot;
        int i;
        top = c + size - 1;
        bot = a + size - 1 - digits;

        for (i = 0; i < size - digits; i++, top--, bot--) {
                *top = *bot;
        }
        for (i = 0; i < digits; i++, c++) {
                *c = 0;
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t bn_lshb_low(dig_t *c, const dig_t *a, int size, int bits) {
        int i;
        dig_t r, carry, shift, mask;

//        printf("now in bn_lshb_low...\n");
//        printf("bits: %d ...\n", bits);
        shift = RLC_DIG - bits;
//       printf("a: %" PRIu64 "\n", *a); 
//       printf("shift: %" PRIu64 "\n", shift);

        carry = 0;
        mask = RLC_MASK(bits);
//      printf("mask: %" PRIu64 "\n", mask);

        for (i = 0; i < size; i++, a++, c++) {
                /* Get the needed least significant bits. */
                r = ((*a) >> shift) & mask;
                /* Shift left the operand. */
                *c = ((*a) << bits) | carry;

//                printf("c: %" PRIu64 "\n", *c);
 

                /* Update the carry. */
                carry = r;
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_trim(bn_t a) {
        if (a->used <= a->alloc) {
                while (a->used > 0 && a->dp[a->used - 1] == 0) {
                        --(a->used);
                }
                /* Zero can't be negative. */
                if (a->used <= 0) {
                        a->used = 1;
                        a->dp[0] = 0;
                        a->sign = RLC_POS;
                }
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_lsh(bn_t c, const bn_t a, int bits) {
        int digits;
        dig_t carry;
        bn_copy(c, a);
        if (bits <= 0) {
                return;
        }
        RLC_RIP(bits, digits, bits);
        bn_grow(c, c->used + digits + (bits > 0));
        c->used = a->used + digits;
        c->sign = a->sign;
        if (digits > 0) {
         dv_lshd(c->dp, a->dp, c->used, digits);
        }
        if (bits > 0) {
         if (c != a) {
          carry = bn_lshb_low(c->dp + digits, a->dp, a->used, bits);
         } else {
         carry = bn_lshb_low(c->dp + digits, c->dp + digits, c->used - digits, bits);
         }
          if (carry != 0) {
           c->dp[c->used] = carry;
           (c->used)++;
          }
         }
         bn_trim(c);
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t fp_addn_low(dig_t *c, const dig_t *a, const dig_t *b) {
        int i;
        dig_t carry, c0, c1, r0, r1;
        carry = 0;
        for (i = 0; i < RLC_FP_DIGS; i++, a++, b++, c++) {
                r0 = (*a) + (*b);
                c0 = (r0 < (*a));
                r1 = r0 + carry;
                c1 = (r1 < r0);
                carry = c0 | c1;
                (*c) = r1;
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_subm_low(dig_t *c, const dig_t *a, const dig_t *b) {
        int i;
        dig_t carry, r0, diff;

        /* Zero the carry. */
        carry = 0;
        for (i = 0; i < RLC_FP_DIGS; i++, a++, b++) {
                diff = (*a) - (*b);
                r0 = diff - carry;
                carry = ((*a) < (*b)) || (carry && !diff);
                c[i] = r0;
        }
        if (carry) {
                fp_addn_low(c, c, shared_prime);
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_sub(fp_t c, const fp_t a, const fp_t b) {
        fp_subm_low(c, a, b);
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t fp_subn_low(dig_t *c, const dig_t *a, const dig_t *b) {
        int i;
        dig_t carry, r0, diff;

      //  printf("calling fp_subn_low...\n");
        /* Zero the carry. */
        carry = 0;
        for (i = 0; i < RLC_FP_DIGS; i++, a++, b++, c++) {
                diff = (*a) - (*b);
                r0 = diff - carry;
                carry = ((*a) < (*b)) || (carry && !diff);
                (*c) = r0;
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
int dv_cmp(const dig_t *a, const dig_t *b, int size) {
        int i, r;
        a += (size - 1);
        b += (size - 1);
        r = RLC_EQ;
        for (i = 0; i < size; i++, --a, --b) {
                if (*a != *b && r == RLC_EQ) {
                        r = (*a > *b ? RLC_GT : RLC_LT);
                }
        }
        return r;
}
__device__
#if INLINE == 0
__noinline__
#endif
void dv_zero(dig_t *a, int digits) {
        int i;
        for (i = 0; i < digits; i++, a++) {
                (*a) = 0;
        }
        return;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_zero(bn_t a) {
        a->sign = RLC_POS;
        a->used = 1;
        dv_zero(a->dp, a->alloc);
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_set_dig(bn_t a, dig_t digit) {
        bn_zero(a);
        a->dp[0] = digit;
        a->used = 1;
        a->sign = RLC_POS;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_mod_pre_monty(bn_t u, const bn_t m) {
// Even though the algorithm works for any r which is relatively prime to n, it is more useful when r is taken to be a power of 2. In this case, the Montgomery algorithm performs divisions by a power of 2, which is an intrinsically fast operation on general-purpose computers, e.g., signal processors and microprocessors; this leads to a simpler implementation than ordinary modular multiplication, which is typically faster as well [7].

 dig_t x, b;
 b = m->dp[0];

 if ((b & 0x01) == 0) {
  printf("+++++++++ Error +++++++++ \n");
  printf("(b & 0x01) == 0 in bn_mod_pre_monty ...\n");
  printf("++++++++ !Error! ++++++++++ \n");
  return;
 }

 x = (((b + 2) & 4) << 1) + b;                           /* here x*a==1 mod 2**4 */
 x *= (dig_t)2 - b * x;                                          /* here x*a==1 mod 2**8 */
 x *= (dig_t)2 - b * x;                                          /* here x*a==1 mod 2**16 */
 x *= (dig_t)2 - b * x;                                          /* here x*a==1 mod 2**32 */
 x *= (dig_t)2 - b * x;                                          /* here x*a==1 mod 2**64 */
 /* u = -1/m0 (mod 2^RLC_DIG) */
 bn_set_dig(u, -x);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_rdcn_low(dig_t *c, dig_t *a) {
 int i, j;
 dig_t t, r0, r1, r2, *tmp, *tmpc, u;
 const dig_t *tmpm, *m;
 //   u = *(fp_prime_get_rdc());
 //   m = fp_prime_get();
 u = *shared_u;
 m = shared_prime;

// printf("u in fp_rdcn_low: \n");
// print_multiple_precision(&u,6);
// print_line();
//
// printf("m in fp_rdcn_low: \n");
// print_multiple_precision(m,6);
// print_line();

 printf("a in fp_rdcn_low: \n");
 print_multiple_precision(a,6);
 print_line();

 tmpc = c;
 r0 = r1 = r2 = 0;
 for (i = 0; i < RLC_FP_DIGS; i++, tmpc++, a++) {
  tmp = c;
  tmpm = m + i;
  for (j = 0; j < i; j++, tmp++, tmpm--) {
   RLC_COMBA_STEP_MUL(r2, r1, r0, *tmp, *tmpm);
  }
  RLC_COMBA_ADD(t, r2, r1, r0, *a);
  *tmpc = (dig_t)(r0 * u);
  RLC_COMBA_STEP_MUL(r2, r1, r0, *tmpc, *m);
  r0 = r1;
  r1 = r2;
  r2 = 0;
 }
 for (i = RLC_FP_DIGS; i < 2 * RLC_FP_DIGS - 1; i++, a++) {
  tmp = c + (i - RLC_FP_DIGS + 1);
  tmpm = m + RLC_FP_DIGS - 1;
  for (j = i - RLC_FP_DIGS + 1; j < RLC_FP_DIGS; j++, tmp++, tmpm--) {
   RLC_COMBA_STEP_MUL(r2, r1, r0, *tmp, *tmpm);
  }
  RLC_COMBA_ADD(t, r2, r1, r0, *a);
  c[i - RLC_FP_DIGS] = r0;
  r0 = r1;
  r1 = r2;
  r2 = 0;
  }
  RLC_COMBA_ADD(t, r2, r1, r0, *a);
  c[RLC_FP_DIGS - 1] = r0;
  if (r1 || dv_cmp(c, m, RLC_FP_DIGS) != RLC_LT) {
   fp_subn_low(c, c, m);
  }
}
// Function to multiply two integers
__device__
#if INLINE == 0
__noinline__
#endif
void fp_muln_low(dig_t *c, const dig_t *a, const dig_t *b) {
        int i, j;
        const dig_t *tmpa, *tmpb;
        dig_t r0, r1, r2;
// printf("input in fp_muln_low...\n ");
// printf("a: %" PRIu64 "\n", *a);
// printf("b: %" PRIu64 "\n", *b);
        r0 = r1 = r2 = 0;
        for (i = 0; i < RLC_FP_DIGS; i++, c++) {
                tmpa = a;
                tmpb = b + i;
                for (j = 0; j <= i; j++, tmpa++, tmpb--) {
                        RLC_COMBA_STEP_MUL(r2, r1, r0, *tmpa, *tmpb);
                }
//                printf("1. setting r0 to: %" PRIu64 "\n", r0);
                *c = r0;
                r0 = r1;
                r1 = r2;
                r2 = 0;
        }
        for (i = 0; i < RLC_FP_DIGS; i++, c++) {
                tmpa = a + i + 1;
                tmpb = b + (RLC_FP_DIGS - 1);
                for (j = 0; j < RLC_FP_DIGS - (i + 1); j++, tmpa++, tmpb--) {
                        RLC_COMBA_STEP_MUL(r2, r1, r0, *tmpa, *tmpb);
                }
//                printf("2. setting r0 to: %" PRIu64 "\n", r0);
                *c = r0;
                r0 = r1;
                r1 = r2;
                r2 = 0;
        }
// printf("result at fp_muln_low: %" PRIu64 "\n", *c);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_mulm_low(dig_t *c, const dig_t *a, const dig_t *b) {
// The results has more bits then the operands...
        dig_t *t;
        t = (dig_t* ) malloc(2 * RLC_FP_DIGS * sizeof(dig_t));
// Multiply the two numbers
        fp_muln_low(t, a, b);
// Reduce the result using c
        fp_rdcn_low(c, t);
        free(t);
}
__device__
#if INLINE == 0
__noinline__
#endif
uint64_t bn_rshb_low(uint64_t *c, const uint64_t *a, int size, int bits) {
        int i;
        uint64_t r, carry, shift, mask;
        c += size - 1;
        a += size - 1;
        /* Prepare the bit mask. */
        shift = (RLC_DIG - bits) % RLC_DIG;
        carry = 0;
        mask = RLC_MASK(bits);
        for (i = size - 1; i >= 0; i--, a--, c--) {
                /* Get the needed least significant bits. */
                r = (*a) & mask;
                /* Shift left the operand. */
                *c = ((*a) >> bits) | (carry << shift);
                /* Update the carry. */
                carry = r;
        }
        return carry;
}

__device__
#if INLINE == 0
__noinline__
#endif
uint64_t add_multiple_precision_1(uint64_t *c, const uint64_t *a, uint64_t digit, int size) {
        int i;
        register uint64_t carry, r0;

        carry = digit;
        for (i = 0; i < size && carry; i++, a++, c++) {
                r0 = (*a) + carry;
                carry = (r0 < carry);
                (*c) = r0;
        }
        for (; i < size; i++, a++, c++) {
                (*c) = (*a);
        }
        return carry;
}

__device__
#if INLINE == 0
__noinline__
#endif
uint64_t add_multiple_precision(uint64_t *c, const uint64_t *a, const uint64_t *b, int size) {
        int i;
        register uint64_t carry, c0, c1, r0, r1;

        carry = 0;
        for (i = 0; i < size; i++, a++, b++, c++) {
                r0 = (*a) + (*b);
                c0 = (r0 < (*a));
                r1 = r0 + carry;
                c1 = (r1 < r0);
                carry = c0 | c1;
                (*c) = r1;
        }
        return carry;
}

__device__
#if INLINE == 0
__noinline__
#endif
uint64_t subtract_multiple_precision_1(uint64_t *c, const uint64_t *a, uint64_t digit, int size) {
        int i;
        uint64_t carry, r0;

        carry = digit;
        for (i = 0; i < size && carry; i++, c++, a++) {
                r0 = (*a) - carry;
                carry = (r0 > (*a));
                (*c) = r0;
        }
        for (; i < size; i++, a++, c++) {
                (*c) = (*a);
        }
        return carry;
}

__device__
#if INLINE == 0
__noinline__
#endif
uint64_t multiply_multiple_precision(uint64_t *c, const uint64_t *a, uint64_t digit, int size) {
        uint64_t r0, r1, carry = 0;
        for (int i = 0; i < size; i++, a++, c++) {
                r1 = ((__uint128_t)(*a) * (__uint128_t)(digit)) >> 64;
                r0 = ( *a) * (digit); 
                *c = r0 + carry;
                carry = r1 + (*c < carry);
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void right_shift(uint64_t *c, const uint64_t *a, int size, int digits) {
        const uint64_t *top;
        uint64_t *bot;
        int i;

        top = a + digits;
        bot = c;

        for (i = 0; i < size - digits; i++, top++, bot++) {
                *bot = *top;
        }
        for (; i < size; i++, bot++) {
                *bot = 0;
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void left_shift(uint64_t *result, const uint64_t *input, int size, int digits){
// This function left shifts bits
//        uint64_t *top;
//        const dig_t *bot;
//        int i;
//
//        top = c + size - 1;
//        bot = a + size - 1 - digits;
//
//        for (i = 0; i < size - digits; i++, top--, bot--) {
//                *top = *bot;
//        }
//        for (i = 0; i < digits; i++, c++) {
//                *c = 0;
//        }
 uint64_t *top;
 const uint64_t *bot;
 top = result + size - 1;
 bot = input + size - 1 - digits;
 for (int i = 0; i < size - digits; i++, top--, bot--) {
         *top = *bot;
 }
 for (int i = 0; i < digits; i++, result++) {
         *result = 0;
 }
}
__device__
#if INLINE == 0
__noinline__
#endif
uint64_t shift_bits(uint64_t *result, const uint64_t *input, int size, int bits){
// This function shifts bits

//         int i;
//         dig_t r, carry, shift, mask;
//                 
//         shift = RLC_DIG - bits; 
//         carry = 0;
//         mask = RLC_MASK(bits);
// 
//         for (i = 0; i < size; i++, a++, c++) {
//                 /* Get the needed least significant bits. */
//                 r = ((*a) >> shift) & mask;
//                 /* Shift left the operand. */
//                 *c = ((*a) << bits) | carry;
//                 /* Update the carry. */
//                 carry = r;
//         }
//         return carry;
 uint64_t r, carry, shift, mask;
 
/**
 * Size in bits of a digit.
 */
 shift = 64 - bits;
 carry = 0;
 if(bits > 64){
  printf("shift_bits cannot shift this much, exciting...\n");
  return;
 }
 mask = (((uint64_t)1 << ((bits) % 64)) - 1);
 for (int i = 0; i < size; i++, input++, result++) {
         r = ((*input) >> shift) & mask;
         *result = ((*input) << bits) | carry;
         carry = r;
 }
 return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
int compare_multiple_precision(uint64_t *a, uint64_t *b, int size){
// Returns 1 if a > b, returns 0 if a == b, returns 2 if b > a
        int i, r;

        a += (size - 1);
        b += (size - 1);

        r = 0;
        for (i = 0; i < size; i++, --a, --b) {
                if (*a != *b && r == 0) {
                        r = (*a > *b ? 1 : 2);
                }
        }
        return r;
}
__device__
#if INLINE == 0
__noinline__
#endif
int subtract_multiple_precision(uint64_t *c, const uint64_t *a, const uint64_t *b, int size){
// a - b
        int i;
        uint64_t carry, r0, diff;

        /* Zero the carry. */
        carry = 0;
        for (i = 0; i < size; i++, a++, b++, c++) {
                diff = (*a) - (*b);
                r0 = diff - carry;
                carry = ((*a) < (*b)) || (carry && !diff);
                (*c) = r0;
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t bn_subn_low(dig_t *c, const dig_t *a, const dig_t *b, int size) {
        int i;
        dig_t carry, r0, diff;

        /* Zero the carry. */
        carry = 0;
        for (i = 0; i < size; i++, a++, b++, c++) {
                diff = (*a) - (*b);
                r0 = diff - carry;
                carry = ((*a) < (*b)) || (carry && !diff);
                (*c) = r0;
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_modn_low(dig_t *c, const dig_t *a, int sa, const dig_t *m, int sm, dig_t u) {
        int i, j;
        dig_t t, r0, r1, r2;
        dig_t *tmp, *tmpc;
        const dig_t *tmpm;

        tmpc = c;

        r0 = r1 = r2 = 0;
        for (i = 0; i < sm; i++, tmpc++, a++) {
                tmp = c;
                tmpm = m + i;
                for (j = 0; j < i; j++, tmp++, tmpm--) {
                        RLC_COMBA_STEP_MUL(r2, r1, r0, *tmp, *tmpm);
                }
                if (i < sa) {
                        RLC_COMBA_ADD(t, r2, r1, r0, *a);
                }
                *tmpc = (dig_t)(r0 * u);
                RLC_COMBA_STEP_MUL(r2, r1, r0, *tmpc, *m);
                r0 = r1;
                r1 = r2;
                r2 = 0;
        }
        for (i = sm; i < 2 * sm - 1; i++, a++) {
                tmp = c + (i - sm + 1);
                tmpm = m + sm - 1;
                for (j = i - sm + 1; j < sm; j++, tmp++, tmpm--) {
                        RLC_COMBA_STEP_MUL(r2, r1, r0, *tmp, *tmpm);
                }
                if (i < sa) {
                        RLC_COMBA_ADD(t, r2, r1, r0, *a);
                }
                c[i - sm] = r0;
                r0 = r1;
                r1 = r2;
                r2 = 0;
        }

        if (i < sa) {
                RLC_COMBA_ADD(t, r2, r1, r0, *a);
        }
        c[sm - 1] = r0;
        if (r1) {
                bn_subn_low(c, c, m, sm);
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
int bn_is_zero(const bn_t a) {
        if (a->used == 0) {
                return 1;
        }
        if ((a->used == 1) && (a->dp[0] == 0)) {
                return 1;
        }
        return 0;
}
__device__
#if INLINE == 0
__noinline__
#endif
int bn_cmp_abs(const bn_t a, const bn_t b) {

//        printf("Inside bn_cmp_abs...\n");
//        printf("a-> used %d ....\n", a->used);
//        printf("b-> used %d ....\n", b->used);

        if (bn_is_zero(a) && bn_is_zero(b)) {
                return RLC_EQ;
        }

        if (a->used > b->used) {
                return RLC_GT;
        }

        if (a->used < b->used) {
                return RLC_LT;
        }

        return dv_cmp(a->dp, b->dp, a->used);
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_make(bn_t a, int digits) {
        if (digits < 0) {
                printf("digits < 0 in bn_make...");
        }
        /* Allocate at least one digit. */
        digits = RLC_MAX(digits, 1);
        if (a != NULL) {
                a->used = 1;
                a->dp[0] = 0;
                a->alloc = digits;
                a->sign = RLC_POS;
        }
}
#define bn_new_size(A, D)    \
        bn_make(A, D);       \

__device__
#if INLINE == 0
__noinline__
#endif
void bn_new(bn_t t){
 bn_make(t, RLC_BN_SIZE);
}

// __device__
#if INLINE == 0
__noinline__
#endif
// void bn_mod_monty_comba(bn_t c, const bn_t a, const bn_t m, const bn_t u) {
//         int digits; 
//         bn_t t; 
// 
//         printf(" now reducing the converted message using combat monty... \n");
//         digits = 2 * m->used;
//         bn_new_size(t, digits);
//         bn_zero(t);
// 
//         bn_modn_low(t->dp, a->dp, a->used, m->dp, m->used, u->dp[0]);
//         t->used = m->used;
// 
//         bn_trim(t);
//         if (bn_cmp_abs(t, m) != RLC_LT) {
//          bn_sub(t, t, m);
//         }
//         bn_copy(c, t);
// }

__device__
#if INLINE == 0
__noinline__
#endif
void bn_abs(bn_t c, const bn_t a) {
        if (c->dp != a->dp) {
                bn_copy(c, a);
        }
        c->sign = RLC_POS;
}
__device__
#if INLINE == 0
__noinline__
#endif
int bn_sign(const bn_t a) {
        return a->sign;
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t bn_addn_low(dig_t *c, const dig_t *a, const dig_t *b, int size) {
        int i;
        register dig_t carry, c0, c1, r0, r1;

        carry = 0;
        for (i = 0; i < size; i++, a++, b++, c++) {
                r0 = (*a) + (*b);
                c0 = (r0 < (*a));
                r1 = r0 + carry;
                c1 = (r1 < r0);
                carry = c0 | c1;
                (*c) = r1;
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t bn_add1_low(dig_t *c, const dig_t *a, dig_t digit, int size) {
        int i;
        register dig_t carry, r0;

        carry = digit;
        for (i = 0; i < size && carry; i++, a++, c++) {
                r0 = (*a) + carry;
                carry = (r0 < carry);
                (*c) = r0;
        }
        for (; i < size; i++, a++, c++) {
                (*c) = (*a);
        }
        return carry;
}
/**
 * Adds two multiple precision integers, where a >= b.
 *
 * @param[out] c        - the result.
 * @param[in] a         - the first multiple precision integer to add.
 * @param[in] b         - the second multiple precision integer to add.
 */
__device__
#if INLINE == 0
__noinline__
#endif
static void bn_add_imp(bn_t c, const bn_t a, const bn_t b) {
        int max, min;
        dig_t carry;

        max = a->used;
        min = b->used;

        if (min == 0) {
                bn_copy(c, a);
                return;
        }
                /* Grow the result. */
                bn_grow(c, max);

                if (a->used == b->used) {
                        carry = bn_addn_low(c->dp, a->dp, b->dp, max);
                } else {
                        carry = bn_addn_low(c->dp, a->dp, b->dp, min);
                        carry = bn_add1_low(c->dp + min, a->dp + min, carry, max - min);
                }
                if (carry) {
                        bn_grow(c, max + 1);
                        c->dp[max] = carry;
                }
                c->used = max + carry;
                bn_trim(c);
}

__device__
#if INLINE == 0
__noinline__
#endif
dig_t bn_sub1_low(dig_t *c, const dig_t *a, dig_t digit, int size) {
        int i;
        dig_t carry, r0;

        carry = digit;
        for (i = 0; i < size && carry; i++, c++, a++) {
                r0 = (*a) - carry;
                carry = (r0 > (*a));
                (*c) = r0;
        }
        for (; i < size; i++, a++, c++) {
                (*c) = (*a);
        }
        return carry;
}


/**
 * Subtracts two multiple precision integers, where a >= b.
 *
 * @param[out] c        - the result.
 * @param[in] a         - the first multiple precision integer to subtract.
 * @param[in] b         - the second multiple precision integer to subtract.
 */
__device__
#if INLINE == 0
__noinline__
#endif
static void bn_sub_imp(bn_t c, const bn_t a, const bn_t b) {
        int max, min;
        dig_t carry;

        max = a->used;
        min = b->used;

        if (min == 0) {
                bn_copy(c, a);
                return;
        }

                /* Grow the destination to accomodate the result. */
                bn_grow(c, max);

                if (a->used == b->used) {
                        carry = bn_subn_low(c->dp, a->dp, b->dp, min);
                } else {
                        carry = bn_subn_low(c->dp, a->dp, b->dp, min);
                        carry = bn_sub1_low(c->dp + min, a->dp + min, carry, max - min);
                }
                c->used = max;
                bn_trim(c);
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_sub(bn_t c, const bn_t a, const bn_t b) {
        int sa, sb;

        sa = a->sign;
        sb = b->sign;

        if (sa != sb) {
                /* If the signs are different, copy the sign of the first number and
                 * add. */
                c->sign = sa;
                if (bn_cmp_abs(a, b) == RLC_LT) {
                        bn_add_imp(c, b, a);
                } else {
                        bn_add_imp(c, a, b);
                }
        } else {
                /* If the signs are equal, adjust the sign and subtract. */
                if (bn_cmp_abs(a, b) != RLC_LT) {
                        bn_sub_imp(c, a, b);
                        c->sign = sa;
                } else {
                        bn_sub_imp(c, b, a);
                        c->sign = (sa == RLC_POS) ? RLC_NEG : RLC_POS;
                }
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_neg(bn_t c, const bn_t a) {
        if (c->dp != a->dp) {
                bn_copy(c, a);
        }
        if (!bn_is_zero(c)) {
                c->sign = a->sign ^ 1;
        }
}

__device__
#if INLINE == 0
__noinline__
#endif
void bn_add(bn_t c, const bn_t a, const bn_t b) {
        int sa, sb;

        sa = a->sign;
        sb = b->sign;

        if (sa == sb) {
                /* If the signs are equal, copy the sign and add. */
                c->sign = sa;
                if (bn_cmp_abs(a, b) == RLC_LT) {
                        bn_add_imp(c, b, a);
                } else {
                        bn_add_imp(c, a, b);
                }
        } else {
                /* If the signs are different, subtract. */
                if (bn_cmp_abs(a, b) == RLC_LT) {
                        bn_sub_imp(c, b, a);
                        c->sign = sb;
                } else {
                        bn_sub_imp(c, a, b);
                        c->sign = sa;
                }
        }
}

__device__
#if INLINE == 0
__noinline__
#endif
void dv_rshd(dig_t *c, const dig_t *a, int size, int digits) {
        const dig_t *top;
        dig_t *bot;
        int i;
    
        top = a + digits;
        bot = c;

        for (i = 0; i < size - digits; i++, top++, bot++) {
                *bot = *top;
        }
        for (; i < size; i++, bot++) {
                *bot = 0;
        }
}
#define RLC_DIV_DIG(Q, R, H, L, D)                                                                                      \
        Q = (((dbl_t)(H) << RLC_DIG) | (L)) / (D);                                                              \
        R = (((dbl_t)(H) << RLC_DIG) | (L)) - (dbl_t)(Q) * (dbl_t)(D);                  \
 
__device__
#if INLINE == 0
__noinline__
#endif
dig_t bn_mul1_low(dig_t *c, const dig_t *a, dig_t digit, int size) {
        dig_t r0, r1, carry = 0;
        for (int i = 0; i < size; i++, a++, c++) {
                RLC_MUL_DIG(r1, r0, *a, digit);
                *c = r0 + carry;
                carry = r1 + (*c < carry);
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_divn_low(dig_t *c, dig_t *d, dig_t *a, int sa, dig_t *b, int sb) {
	int norm, i, n, t, sd;
	dig_t carry, t1[3], t2[3];

	/* Normalize x and y so that the leading digit of y is bigger than
	 * 2^(RLC_DIG-1). */


	norm = util_bits_dig(b[sb - 1]) % RLC_DIG;
//        printf("\n.B 1 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }

	if (norm < (int)(RLC_DIG - 1)) {
		norm = (RLC_DIG - 1) - norm;
		carry = bn_lshb_low(a, a, sa, norm);
		if (carry) {
			a[sa++] = carry;
		}

//        printf("\n.B 2 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }
		carry = bn_lshb_low(b, b, sb, norm);
//        printf("\n.B 3 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }


		if (carry) {
			b[sb++] = carry;
		}
	} else {
		norm = 0;
	}
//        exit(0);

	n = sa - 1;
	t = sb - 1;

//        printf("\n.B 4 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }


	/* Shift y so that the most significant digit of y is aligned with the
	 * most significant digit of x. */
	dv_lshd(b, b, sb + (n - t), (n - t));
//        printf("\n.B 5 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }


//	gmp_printf ("b %Mu\n", b[0]);
//        gmp_printf ("b %Mu\n", b[1]);
//        gmp_printf ("b %Mu\n", b[2]);
//        gmp_printf ("b %Mu\n", b[3]);
//        gmp_printf ("b %Mu\n", b[4]);
//        gmp_printf ("b %Mu\n", b[5]);
//        printf("\n....................... \n");

	/* Find the most significant digit of the quotient. */
//        printf("Let us enter the loop...\n");
	while (dv_cmp(a, b, sa) != RLC_LT) {
		c[n - t]++;
//                printf("arithmetic call...\n");
//                printf("%" PRIu64 "\n", c[n - t]);
		bn_subn_low(a, a, b, sa);
	}

//        printf("\n.B 6 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }

	/* Shift y back. */
	dv_rshd(b, b, sb + (n - t), (n - t));
//        printf("\n.B 7 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }


	/* Find the remaining digits. */
//        printf("limits: %d %d ", n, t+1);
	for (i = n; i >= (t + 1); i--) {
//        printf("t equlas: %d \n", t);
		dig_t tmp;

		if (i > sa) {
			continue;
		}

		if (a[i] == b[t]) {
			c[i - t - 1] = RLC_MASK(RLC_DIG);
		} else {
			RLC_DIV_DIG(c[i - t - 1], tmp, a[i], a[i - 1], b[t]);
		}
//        printf("\n.B 8 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }

		c[i - t - 1]++;
		do {
			c[i - t - 1]--;
			t1[0] = (t - 1 < 0) ? 0 : b[t - 1];
			t1[1] = b[t];

			carry = bn_mul1_low(t1, t1, c[i - t - 1], 2);
			t1[2] = carry;

			t2[0] = (i - 2 < 0) ? 0 : a[i - 2];
			t2[1] = (i - 1 < 0) ? 0 : a[i - 1];
			t2[2] = a[i];
		} while (dv_cmp(t1, t2, 3) == RLC_GT);
//        printf("\n.B 9 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }


		carry = bn_mul1_low(d, b, c[i - t - 1], sb);
		sd = sb;
		if (carry) {
			d[sd++] = carry;
		}

		carry = bn_subn_low(a + (i - t - 1), a + (i - t - 1), d, sd);
		sd += (i - t - 1);
		if (sa - sd > 0) {
			carry = bn_sub1_low(a + sd, a + sd, carry, sa - sd);
		}

//        printf("\n.B 10 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }

		if (carry) {
			sd = sb + (i - t - 1);
			carry = bn_addn_low(a + (i - t - 1), a + (i - t - 1), b, sb);
			carry = bn_add1_low(a + sd, a + sd, carry, sa - sd);
			c[i - t - 1]--;
		}
	}
	/* Remainder should be not be longer than the divisor. */
//        printf("\n.B 11 ..\n");
//        for(int i=0; i < sb; i++){
//         printf ("b%d: %" PRIu64 "\n",i,  b[i]);
//        }

	bn_rshb_low(d, a, sb, norm);

}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_sub_dig(bn_t c, const bn_t a, dig_t b) {
        dig_t carry;

                bn_grow(c, a->used);

                /* If a < 0, compute c = -(|a| + b). */
                if (a->sign == RLC_NEG) {
                        carry = bn_add1_low(c->dp, a->dp, b, a->used);
                        if (carry) {
                                bn_grow(c, a->used + 1);
                                c->dp[a->used] = carry;
                        }
                        c->used = a->used + carry;
                        c->sign = RLC_NEG; 
                } else {
                        /* If a > 0 && |a| >= b, compute c = (|a| - b). */
                        if (a->used > 1 || a->dp[0] >= b) {
                                carry = bn_sub1_low(c->dp, a->dp, b, a->used);
                                c->used = a->used;
                                c->sign = RLC_POS;
                        } else {
                                /* If a > 0 && a < b. */
                                if (a->used == 1) {
                                        c->dp[0] = b - a->dp[0];
                                } else {
                                        c->dp[0] = b;
                                }
                                c->used = 1;
                                c->sign = RLC_NEG;
                        }
                }
                bn_trim(c);
}

/**
 * Divides two multiple precision integers, computing the quotient and the
 * remainder.
 *
 * @param[out] c                - the quotient.
 * @param[out] d                - the remainder.
 * @param[in] a                 - the dividend.
 * @param[in] b                 - the the divisor.
 */
__device__
#if INLINE == 0
__noinline__
#endif
void bn_div_imp(bn_t c, bn_t d, const bn_t a, const bn_t b) {
        bn_t q, x, y, r;
        int sign;

// printf("1. bn_div_imp");
        x = (bn_t) malloc(sizeof(bn_st));
// printf("2. bn_div_imp");
        x->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
// printf("3. bn_div_imp");
        x->alloc = RLC_BN_SIZE;
// printf("4. bn_div_imp");
        x->sign = RLC_POS;

        q = (bn_t) malloc(sizeof(bn_st));
// printf("5. bn_div_imp");
        q->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
// printf("6. bn_div_imp");
        q->alloc = RLC_BN_SIZE;
// printf("7. bn_div_imp");
        y = (bn_t) malloc(sizeof(bn_st));
// printf("8. bn_div_imp");
        y->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
// printf("9. bn_div_imp");
        y->alloc = RLC_BN_SIZE;
// printf("10. bn_div_imp");
        y->sign = RLC_POS;

        r = (bn_t) malloc(sizeof(bn_st));
// printf("11. bn_div_imp");
        r->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
// printf("12. bn_div_imp");
        r->alloc = RLC_BN_SIZE;
// printf("13. bn_div_imp");
        r->sign = RLC_POS;

        bn_new(x);
        bn_new(q);
        bn_new(y);
        bn_new(r);

//        printf("a-> used %d ....\n", a->used);
//        printf("b-> used %d ....\n", b->used);

//	printf ("a1: %" PRIu64 "\n", a->dp[0]);
//	printf ("a2: %" PRIu64 "\n", a->dp[1]);
//	printf ("a3: %" PRIu64 "\n", a->dp[2]);
//	printf ("a4: %" PRIu64 "\n", a->dp[3]);
//	printf ("a5: %" PRIu64 "\n", a->dp[4]);
//	printf ("a6: %" PRIu64 "\n", a->dp[5]);
//        printf("\n....................... \n");
//        printf("\n....................... \n");
//        printf("\n....................... \n");
//	printf ("b1: %" PRIu64 "\n", b->dp[0]);
//	printf ("b2: %" PRIu64 "\n", b->dp[1]);
//	printf ("b3: %" PRIu64 "\n", b->dp[2]);
//	printf ("b4: %" PRIu64 "\n", b->dp[3]);
//	printf ("b5: %" PRIu64 "\n", b->dp[4]);
//	printf ("b6: %" PRIu64 "\n", b->dp[5]);

        /* If |a| < |b|, we're done. */
// printf("14. bn_div_imp");
        if (bn_cmp_abs(a, b) == RLC_LT) {

//        printf("bn_cmp_abs(a, b) == RLC_LT...\n");
//        printf("a->sign: %d\n", a->sign);
//        printf("b->sign: %d\n", b->sign);

                if (bn_sign(a) == bn_sign(b)) {
//// printf("15. bn_div_imp");
                        if (c != NULL) {
//                        printf("bn_zero ...\n");
                                bn_zero(c);
                        }
                        if (d != NULL) {
                                bn_copy(d, a);
                        }
                } else {
// printf("16. bn_div_imp");
                        if (c != NULL) {
                                bn_set_dig(c, 1);
                                bn_neg(c, c);
                        }
                        if (d != NULL) {
                                bn_add(d, a, b);
                        }
                }
//                printf("Returning from function call...");
                return;
        }

                /* Be conservative about space for scratch memory, many attempts to
                 * optimize these had invalid reads. */
// printf("17. bn_div_imp");

                bn_new_size(x, a->used + 1);
// printf("18. bn_div_imp");
                bn_new_size(q, a->used + 1);
                bn_new_size(y, a->used + 1);
                bn_new_size(r, a->used + 1);

                bn_zero(q);
                bn_zero(r);
// printf("19. bn_div_imp");
                bn_abs(x, a);
                bn_abs(y, b);

//                printf("calling bn_divn_low...\n");

                /* Find the sign. */
                sign = (a->sign == b->sign ? RLC_POS : RLC_NEG);

// printf("20. bn_div_imp");
                bn_divn_low(q->dp, r->dp, x->dp, a->used, y->dp, b->used);
// printf("21. bn_div_imp");


                q->used = a->used - b->used + 1;
                q->sign = sign;
                bn_trim(q);

                r->used = b->used;
                r->sign = b->sign;
                bn_trim(r);

                /* We have the quotient in q and the remainder in r. */
                if (c != NULL) {
                        if ((bn_is_zero(r)) || (bn_sign(a) == bn_sign(b))) {
                                bn_copy(c, q);
                        } else {
                                bn_sub_dig(c, q, 1);
                        }
                }

                if (d != NULL) {
                        if ((bn_is_zero(r)) || (bn_sign(a) == bn_sign(b))) {
                                bn_copy(d, r);
                        } else {
                                bn_sub(d, b, r);
                        }
                }
//       printf("leaving bn_div_imp...\n");
//        printf("\n....................... \n");
//	printf ("d1: %" PRIu64 "\n", d->dp[0]);
//	printf ("d2: %" PRIu64 "\n", d->dp[1]);
//	printf ("d3: %" PRIu64 "\n", d->dp[2]);
//	printf ("d4: %" PRIu64 "\n", d->dp[3]);
//	printf ("d5: %" PRIu64 "\n", d->dp[4]);
//	printf ("d6: %" PRIu64 "\n", d->dp[5]);
        free(q->dp);
        free(q);
        free(y->dp);
        free(y);
        free(r->dp);
        free(r);
        free(x->dp);
        free(x);
}


__device__
#if INLINE == 0
__noinline__
#endif
void bn_div_rem(bn_t c, bn_t d, const bn_t a, const bn_t b) {
 if (bn_is_zero(b)) {
  printf("bn_div_rem zero!!!...\n");
  return;
 }
 bn_div_imp(c, d, a, b);
}

__device__
#if INLINE == 0
__noinline__
#endif
void bn_mod_basic(bn_t c, const bn_t a, const bn_t m) {
        bn_div_rem(NULL, c, a, m);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_add_basic(fp_t c, const fp_t a, const fp_t b) {
        dig_t carry;

        carry = fp_addn_low(c, a, b);
        if (carry || (dv_cmp(c, shared_prime, RLC_FP_DIGS) != RLC_LT)) {
                carry = fp_subn_low(c, c, shared_prime);
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t fp_mul1_low(dig_t *c, const dig_t *a, dig_t digit) {
        dig_t r0, r1, carry = 0;
        for (int i = 0; i < RLC_FP_DIGS; i++, a++, c++) {
                RLC_MUL_DIG(r1, r0, *a, digit);
                *c = r0 + carry;
                carry = r1 + (*c < carry);
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_zero(fp_t a) {
        dv_zero(a, RLC_FP_DIGS);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_copy(fp_t c, const fp_t a) {

//    printf("fp_copy... RLC_FP_DIGS: %d \n", RLC_FP_DIGS);
//  if( c == NULL){
//    printf("c has problems.... \n");
//   }
//   else{
//    printf("c is OK .... \n");
//   } 
//
//  if( a == NULL){
//    printf("a has problems.... \n");
//   }
//   else{
//    printf("a is OK .... \n");
//   } 
        dv_copy(c, a, RLC_FP_DIGS);
}    
__device__
#if INLINE == 0
__noinline__
#endif
void fp_rdc_basic(fp_t c, dv_t a) {
        dv_t t0, t1, t2, t3;

        t0 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
        t1 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
        t2 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
        t3 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));

//   printf("RLC_DV_DIGS %d , RLC_FP_DIGS %d inside fp_rdc_basic...\n",RLC_DV_DIGS, RLC_FP_DIGS );

//  printf("a in fp_rdc_basic: \n ");
//  printf("%" PRIu64 "\n", *a);

        dv_copy(t2, a, 2 * RLC_FP_DIGS);
//  printf("t2 in fp_rdc_basic: \n ");
//  printf("%" PRIu64 "\n", *t2);
        dv_copy(t3, shared_prime, RLC_FP_DIGS);

//  printf("t3 in fp_rdc_basic: \n ");
//  printf("%" PRIu64 "\n", *t3);
//  printf("t2 in fp_rdc_basic: \n ");
//  printf("%" PRIu64 "\n", *t2);
// itt a t/knek tul kicsi hely van foglalva es tul fogjak cimezni egymast....

        bn_divn_low(t0, t1, t2, 2 * RLC_FP_DIGS, t3, RLC_FP_DIGS);

//  printf("t0 in fp_rdc_basic: \n ");
//  printf("%" PRIu64 "\n", *t0);
//  printf("t1 in fp_rdc_basic: \n ");
//  printf("%" PRIu64 "\n", *t1);
        fp_copy(c, t1);
//  printf("c in fp_rdc_basic: \n ");
//  printf("%" PRIu64 "\n", *c);
//        printf("leaving fp_rdc_basic...\n");
        free(t0);
        free(t1);
        free(t2);
        free(t3);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_prime_conv_dig(fp_t c, dig_t a) {
// dv_t t;
// t = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));

//        ctx_t *ctx = core_get();
//        bn_null(t);
//        dv_new(t);
//// #if FP_RDC == MONTY
//        if (a != 1) {
//         dv_zero(t, 2 * RLC_FP_DIGS + 1);
//         t[RLC_FP_DIGS] = fp_mul1_low(t, shared_conv, a);
//         fp_rdc_basic(c, t);
//        } else {
//         dv_copy(c, shared_one, RLC_FP_DIGS);
//        }
//// #else
//         (void)ctx;
         fp_zero(c);
         c[0] = a;
// #endif
// free(t);
}

__device__
#if INLINE == 0
__noinline__
#endif
void fp_set_dig(fp_t c, dig_t a) {
        fp_prime_conv_dig(c, a);
}    
__device__
#if INLINE == 0
__noinline__
#endif
int bn_bits(const bn_t a) {
        int bits;

        if (bn_is_zero(a)) {
                return 0;
        }

        /* Bits in lower digits. */
        bits = (a->used - 1) * RLC_DIG;

        return bits + util_bits_dig(a->dp[a->used - 1]);
}
__device__
#if INLINE == 0
__noinline__
#endif

dig_t bn_sqra_low(dig_t *c, const dig_t *a, int size) {
	int i;
	dig_t t, c0, c1;

	t = a[0];

	dig_t r0, r1, _r0, _r1, s0, s1, t0, t1;
	/* Accumulate this column with the square of a->dp[i]. */
	RLC_MUL_DIG(_r1, _r0, t, t);
	r0 = _r0 + c[0];
	r1 = _r1 + (r0 < _r0);
	c[0] = r0;

	/* Update the carry. */
	c0 = r1;
	c1 = 0;

	/* Version of the main loop not using double-precision types. */
	for (i = 1; i < size; i++) {
		RLC_MUL_DIG(_r1, _r0, t, a[i]);
		r0 = _r0 + _r0;
		r1 = _r1 + _r1 + (r0 < _r0);

		s0 = r0 + c0;
		s1 = r1 + (s0 < r0);

		t0 = s0 + c[i];
		t1 = s1 + (t0 < s0);
		c[i] = t0;

		/* Accumulate the old delayed carry. */
		c0 = t1 + c1;
		/* Compute the new delayed carry. */
		c1 = (t1 < s1) || (s1 < r1) || (r1 < _r1) || (c0 < c1);
	}

	c[size] += c0;
	c1 += (c[size] < c0);
	return c1;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_sqr_basic(fp_t c, const fp_t a) {
        int i;
        dv_t t;

        t = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
//  printf("entering  fp_sqr_basic \n");
//  printf("(RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t): %d\n",(RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
//
//  printf("RLC_DV_DIGS  %d\n",RLC_DV_DIGS );
//  printf("FP_PRIME  %d\n",FP_PRIME );
//  printf("RLC_FP_DIGS  %d\n",RLC_FP_DIGS );
//  printf("RLC_DIG  %d\n",RLC_DIG );
//  printf("RLC_BN_SIZE  %d\n",RLC_BN_SIZE );


//  printf("a in fp_sqr_basic: \n ");
//  printf("%" PRIu64 "\n", *a);
//        dv_null(t);
//        dv_new(t);
        dv_zero(t, 2 * RLC_FP_DIGS);
//  printf("t in fp_sqr_basic: \n ");
//  printf("%" PRIu64 "\n", *t);
        for (i = 0; i < RLC_FP_DIGS - 1; i++) {
                t[RLC_FP_DIGS + i + 1] =
                                bn_sqra_low(t + 2 * i, a + i, RLC_FP_DIGS - i);
        }
//  printf("2. t in fp_sqr_basic: \n ");
//  printf("%" PRIu64 "\n", *t);
        bn_sqra_low(t + 2 * i, a + i, 1);
//  printf("3. t in fp_sqr_basic: \n ");
//  printf("%" PRIu64 "\n", *t);
//  printf("calling fp_rdc_basic... \n ");
        fp_rdc_basic(c, t);
//  printf("4. c in fp_sqr_basic: \n ");
//  printf("%" PRIu64 "\n", *c);
        free(t);
//        printf("leaving  fp_sqr_basic \n");
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_sqr(fp_t c, const fp_t a) {
  fp_sqr_basic(c,a);
}
__device__
#if INLINE == 0
__noinline__
#endif
int bn_get_bit(const bn_t a, int bit) {
        int d;
        
        if (bit < 0) {
                printf("bn_get_bit error...\n");
                return 0;
        }

        if (bit > bn_bits(a)) {
                return 0;
        }

        RLC_RIP(bit, d, bit);

        if (d >= a->used) {
                return 0;
        } else {
                return (a->dp[d] >> bit) & (dig_t)1;
        }
}  
__device__
#if INLINE == 0
__noinline__
#endif
dig_t fp_mula_low(dig_t *c, const dig_t *a, dig_t digit) {
        dig_t _c, r0, r1, carry = 0;
        for (int i = 0; i < RLC_FP_DIGS; i++, a++, c++) {
                /* Multiply the digit *a by d and accumulate with the previous
                 * result in the same columns and the propagated carry. */
                RLC_MUL_DIG(r1, r0, *a, digit);
//                printf("%d. : %" PRIu64 "\n",i, *a);
                _c = r0 + carry;
                carry = r1 + (_c < carry);
                /* Increment the column and assign the result. */
                *c = *c + _c;
                /* Update the carry. */
                carry += (*c < _c);
        }
//        printf("returning carry: %" PRIu64 "\n", carry);
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_mul_basic(fp_t c, const fp_t a, const fp_t b) {
        int i;
        dv_t t;
        dig_t carry;

//        dv_null(t);
        /* We need a temporary variable so that c can be a or b. */
//        dv_new(t);
        t = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));

        dv_zero(t, 2 * RLC_FP_DIGS);
        for (i = 0; i < RLC_FP_DIGS; i++) {
                carry = fp_mula_low(t + i, b, *(a + i));
                *(t + i + RLC_FP_DIGS) = carry;
        }
//        printf("result in fp_mul_basic: ");
//        printf("%" PRIu64 "\n", *t);
        fp_rdc_basic(c, t);
//        printf("CCCCC result in fp_mul_basic: ");
//        printf("%" PRIu64 "\n", *c);
        free(t);
}
__device__
#if INLINE == 0
__noinline__
#endif
int fp_is_zero(const fp_t a) {
        int i;
        dig_t t = 0;

//        printf("fp_is_zero called, now iside...\n");

        for (i = 0; i < RLC_FP_DIGS; i++) {
                t |= a[i];
        }
//        printf("leaving function fp_is_zero.\n");
        return !t;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_prime_back(bn_t c, const fp_t a) {
        dv_t t;

        t = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));

        int i;
                bn_grow(c, RLC_FP_DIGS);
                for (i = 0; i < RLC_FP_DIGS; i++) {
                        c->dp[i] = a[i];
                }

#if FP_RDC == MONTY
                dv_zero(t, 2 * RLC_FP_DIGS + 1);
                dv_copy(t, a, RLC_FP_DIGS);
                fp_rdc_basic(c->dp, t);
#endif

                c->used = RLC_FP_DIGS;
                c->sign = RLC_POS;
                bn_trim(c);
                free(t);
}
__device__
#if INLINE == 0
__noinline__
#endif
int bn_cmp_dig(const bn_t a, dig_t b) {
        if (a->sign == RLC_NEG) {
                return RLC_LT;
        }

        if (a->used > 1) {
                return RLC_GT;
        }

        if (a->dp[0] > b) {
                return RLC_GT;
        }

        if (a->dp[0] < b) {
                return RLC_LT;
        }

        return RLC_EQ;
}

__device__
#if INLINE == 0
__noinline__
#endif
dig_t bn_mula_low(dig_t *c, const dig_t *a, dig_t digit, int size) {
        dig_t _c, r0, r1, carry = 0;
        for (int i = 0; i < size; i++, a++, c++) {
                /* Multiply the digit *a by d and accumulate with the previous
                 * result in the same columns and the propagated carry. */
                RLC_MUL_DIG(r1, r0, *a, digit);
                _c = r0 + carry;
                carry = r1 + (_c < carry);
                /* Increment the column and assign the result. */
                *c = *c + _c;
                /* Update the carry. */
                carry += (*c < _c);
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_mul_basic(bn_t c, const bn_t a, const bn_t b) {
        int i;
        bn_t t;

        t  = (bn_t ) malloc(sizeof(bn_st));
        t->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
        t->alloc = RLC_BN_SIZE;
        t->sign = RLC_POS;


        dig_t carry;

                /* We need a temporary variable so that c can be a or b. */
                bn_new_size(t, a->used + b->used);
                bn_zero(t);
                t->used = a->used + b->used;

                for (i = 0; i < a->used; i++) {
                        carry = bn_mula_low(t->dp + i, b->dp, *(a->dp + i), b->used);
                        *(t->dp + i + b->used) = carry;
                }
                t->sign = a->sign ^ b->sign;
                bn_trim(t);

                /* Swap c and t. */
                bn_copy(c, t);
 free(t);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_mul(fp_t c, const fp_t a, const fp_t b) {
 fp_mul_basic(c, a, b);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_prime_conv(fp_t c, const bn_t a) {
 bn_t t; 
 t  = (bn_t ) malloc(sizeof(bn_st));
 t->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 t->alloc = RLC_BN_SIZE;
 t->sign = RLC_POS;

// printf("RLC_BN_SIZE: %d \n ", RLC_BN_SIZE);
// printf("RLC_FP_DIGS: %d \n ", RLC_FP_DIGS);

// printf("input operand a \n ");
// for(int i=0; i < a->used; i++){
//  printf("%" PRIu64 "\n", a->dp[i]);
// }
 /* Reduce a modulo the prime to ensure bounds. */

//                printf("\n printing in fp_prime_conv.... \n");
//                for(int i=0; i < a->used; i++){
//                 printf("a %d %" PRIu64 " \n",i, a->dp[i]);
//                }
 bn_mod_basic(t, a, shared_prime_bn);
//                for(int i=0; i < t->used; i++){
//                 printf("t %d %" PRIu64 " \n",i, t->dp[i]);
//                }
//


 if (bn_is_zero(t)) {
  fp_zero(c);
 } 
 else {
  /* Copy used digits, fill the rest with zero. */
//  printf("t->used: %d \n", t->used);
//  printf("fp_prime_conv printing \n ");
//  for(int i=0; i < t->used; i++){
//   printf("%" PRIu64 "\n", t->dp[i]);
//  }
  dv_copy(c, t->dp, t->used);
  dv_zero(c + t->used, RLC_FP_DIGS - t->used);
// Ezt itt tilos visszakommentezni ha nem MONTY aritmetika van...
//#if FP_RDC == MONTY
//  printf("FP_RDC == MONTY \n ");
// TODO is this ok?
//  printf("Printing c before Montgomery reduction... \n");
//  for(int i=0; i < t->used; i++){
//    printf("c%d %" PRIu64 "\n",i, c[i]);
//  }

//  fp_mul(c, c, &shared_conv[0]);
//
//  printf("Printing c after Montgomery reduction... \n");
//  printf("%" PRIu64 "\n", shared_conv[0]);
//  printf("%" PRIu64 "\n", shared_conv[1]);
//  printf("%" PRIu64 "\n", shared_conv[2]);
//  printf("%" PRIu64 "\n", shared_conv[3]);
//  printf("%" PRIu64 "\n", shared_conv[4]);
//  printf("%" PRIu64 "\n", shared_conv[5]);
//#endif
 }
 free(t->dp);
 free(t);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_inv_exgcd(fp_t c, const fp_t a) {
 bn_t u, v, g1, g2, p, q, r;
 
 u  = (bn_t ) malloc(sizeof(bn_st));
 u->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 u->alloc = RLC_BN_SIZE;
 u->sign = RLC_POS;
 u->used = 1;

 v  = (bn_t ) malloc(sizeof(bn_st));
 v->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 v->alloc = RLC_BN_SIZE;
 v->sign = RLC_POS;
 v->used = 1;

 g1  = (bn_t ) malloc(sizeof(bn_st));
 g1->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 g1->alloc = RLC_BN_SIZE;
 g1->sign = RLC_POS;
 g1->used = 1;

 g2  = (bn_t ) malloc(sizeof(bn_st));
 g2->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 g2->alloc = RLC_BN_SIZE;
 g2->sign = RLC_POS;
 g2->used = 1;

 p  = (bn_t ) malloc(sizeof(bn_st));
 p->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 p->alloc = RLC_BN_SIZE;
 p->sign = RLC_POS;
 p->used = 1;

 q  = (bn_t ) malloc(sizeof(bn_st));
 q->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 q->alloc = RLC_BN_SIZE;
 q->sign = RLC_POS;
 q->used = 1;

 r  = (bn_t ) malloc(sizeof(bn_st));
 r->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 r->alloc = RLC_BN_SIZE;
 r->sign = RLC_POS;
 r->used = 1;

// printf("fp_inv_exgcd called ...\n");
 if (fp_is_zero(a)) {
  printf("fp_is_zero in fp_inv_exgcd...\n");
  return;
 }
 /* u = a, v = p, g1 = 1, g2 = 0. */

       printf("1. a ... \n");
       printf ("a0 %" PRIu64 "\n", *(a) );
       printf ("a1 %" PRIu64 "\n", *(a + 1) );
       printf ("a2 %" PRIu64 "\n", *(a + 2) );
       printf ("a3 %" PRIu64 "\n", *(a + 3) );
       printf ("a4 %" PRIu64 "\n", *(a + 4) );
       printf ("a5 %" PRIu64 "\n", *(a + 5) );

 fp_prime_back(u, a);

       printf("2. u ... \n");
       printf ("u0 %" PRIu64 "\n", *(u->dp) );
       printf ("u1 %" PRIu64 "\n", *(u->dp + 1) );
       printf ("u2 %" PRIu64 "\n", *(u->dp + 2) );
       printf ("u3 %" PRIu64 "\n", *(u->dp + 3) );
       printf ("u4 %" PRIu64 "\n", *(u->dp + 4) );
       printf ("u5 %" PRIu64 "\n", *(u->dp + 5) );

 p->used = RLC_FP_DIGS;
 dv_copy(p->dp, shared_prime, RLC_FP_DIGS);
       printf("3. p ... \n");
       printf ("p0 %" PRIu64 "\n", *(p->dp) );
       printf ("p1 %" PRIu64 "\n", *(p->dp + 1) );
       printf ("p2 %" PRIu64 "\n", *(p->dp + 2) );
       printf ("p3 %" PRIu64 "\n", *(p->dp + 3) );
       printf ("p4 %" PRIu64 "\n", *(p->dp + 4) );
       printf ("p5 %" PRIu64 "\n", *(p->dp + 5) );
 bn_copy(v, p);
 bn_set_dig(g1, 1);
 bn_zero(g2);
 /* While (u != 1. */
 while (bn_cmp_dig(u, 1) != RLC_EQ) {
  /* q = [v/u], r = v mod u. */
  bn_div_rem(q, r, v, u);
       printf("4. r ... \n");
       printf ("r0 %" PRIu64 "\n", *(r->dp) );
       printf ("r1 %" PRIu64 "\n", *(r->dp + 1) );
       printf ("r2 %" PRIu64 "\n", *(r->dp + 2) );
       printf ("r3 %" PRIu64 "\n", *(r->dp + 3) );
       printf ("r4 %" PRIu64 "\n", *(r->dp + 4) );
       printf ("r5 %" PRIu64 "\n", *(r->dp + 5) );

       printf("5. q ... \n");
       printf ("q0 %" PRIu64 "\n", *(q->dp) );
       printf ("q1 %" PRIu64 "\n", *(q->dp + 1) );
       printf ("q2 %" PRIu64 "\n", *(q->dp + 2) );
       printf ("q3 %" PRIu64 "\n", *(q->dp + 3) );
       printf ("q4 %" PRIu64 "\n", *(q->dp + 4) );
       printf ("q5 %" PRIu64 "\n", *(q->dp + 5) );
  /* v = u, u = r. */
  bn_copy(v, u);
  bn_copy(u, r);
  /* r = g2 - q * g1. */
  bn_mul_basic(r, q, g1);
       printf("6. r ... \n");
       printf ("r0 %" PRIu64 "\n", *(r->dp) );
       printf ("r1 %" PRIu64 "\n", *(r->dp + 1) );
       printf ("r2 %" PRIu64 "\n", *(r->dp + 2) );
       printf ("r3 %" PRIu64 "\n", *(r->dp + 3) );
       printf ("r4 %" PRIu64 "\n", *(r->dp + 4) );
       printf ("r5 %" PRIu64 "\n", *(r->dp + 5) );
  bn_sub(r, g2, r);
  /* g2 = g1, g1 = r. */
  bn_copy(g2, g1);
  bn_copy(g1, r);
 }
 if (bn_sign(g1) == RLC_NEG) {
  bn_add(g1, g1, p);
 }
 fp_prime_conv(c, g1);
       printf("7. c ... \n");
       printf ("c0 %" PRIu64 "\n", *(c) );
       printf ("c1 %" PRIu64 "\n", *(c + 1) );
       printf ("c2 %" PRIu64 "\n", *(c + 2) );
       printf ("c3 %" PRIu64 "\n", *(c + 3) );
       printf ("c4 %" PRIu64 "\n", *(c + 4) );
       printf ("c5 %" PRIu64 "\n", *(c + 5) );

 free(g1->dp);
 free(g2->dp);
 free(u->dp);
 free(v->dp);
 free(p->dp);
 free(q->dp);
 free(r->dp);

 free(u);
 free(v);
 free(g1);
 free(g2);
 free(p);
 free(q);
 free(r);

}
// Exponentiates a prime field element. Computes C = A^B (mod p).
__device__
#if INLINE == 0
__noinline__
#endif
void fp_exp_basic(fp_t c, const fp_t a, const bn_t b) {
        int i, l;

//  printf("fp_exp_basic a: \n");
//  print_multiple_precision(a,6);
//  print_line();
//
//  printf("fp_exp_basic b: \n");
//  print_multiple_precision(b->dp,6);
//  print_line();

//  printf("inside fp_exp_basic...\n");
//  printf("a: \n ");
//  printf("%" PRIu64 "\n", *a);
//  printf("b: \n ");
//  printf("%" PRIu64 "\n", b->dp[0]);
// printf("1. fp_exp_basic // ");
        fp_t r;
        r = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
        if(r == NULL){
         printf("r has problems...\n");
        }
        if (bn_is_zero(b)) {
         fp_set_dig(c, 1);
         return;
        }
//        fp_new(r);
        l = bn_bits(b);
        fp_copy(r, a);
        if(r == NULL){
         printf("r has problems...\n");
        }
//  printf("l: %d \n ", l);

//  printf("first r: \n ");
//  printf("%" PRIu64 "\n", *r);

        for (i = l - 2; i >= 0; i--) {
// printf("9. fp_exp_basic // ");
//         printf("1. %d. r %" PRIu64 "\n",i, *r);


         fp_sqr_basic(r, r);
//         printf("2. %d. r %" PRIu64 "\n",i, *r);
//  printf("%d r: \n ", i);
//  printf("%" PRIu64 "\n", *r);

        if(r == NULL){
         printf("r has problems...\n");
        }
// printf("10. fp_exp_basic // ");
// printf("%" PRIu64 "\n", *r);
         if (bn_get_bit(b, i)) {
          fp_mul_basic(r, r, a);
//  printf("fp_mul_basic r: \n ", i);
//  printf("%" PRIu64 "\n", *r);
         }
        }
        if(r == NULL){
         printf("r has problems...\n");
        }
        if (bn_sign(b) == RLC_NEG) {
         fp_inv_exgcd(c, r);
        } else {
// printf("15. fp_exp_basic // ");
        if(r == NULL){
         printf("r has problems...\n");
        }
         fp_copy(c, r);
        }

        free(r);
//  printf("fp_exp_basic c: \n");
//  print_multiple_precision(c,6);
//  print_line();
// printf("16. fp_exp_basic // ");
//  printf("c: \n ");
//  printf("%" PRIu64 "\n", *c);
// printf("leaving fp_exp_basic...\n");
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_inv_basic(fp_t c, const fp_t a) {
        bn_t e;
        e  = (bn_t ) malloc(sizeof(bn_st));
        e->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
        e->alloc = RLC_BN_SIZE;
        e->sign = RLC_POS;

//        bn_null(e);
        if (fp_is_zero(a)) {
                printf("fp_is_zero in fp_inv_basic...\n");
                return;
        }
//                bn_new(e);
        e->used = RLC_FP_DIGS;
        dv_copy(e->dp, shared_prime, RLC_FP_DIGS);
        bn_sub_dig(e, e, 2);
        fp_exp_basic(c, a, e);
        free(e);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_addm_low(dig_t *c, const dig_t *a, const dig_t *b) {
        int i;
        dig_t carry, c0, c1, r0, r1;

        carry = 0;
//        printf("calling fp_addm_low...\n");
//        printf("RLC_FP_DIGS %d ...\n", RLC_FP_DIGS);
        for (i = 0; i < RLC_FP_DIGS; i++, a++, b++) {
                r0 = (*a) + (*b);
                c0 = (r0 < (*a));
                r1 = r0 + carry;
                c1 = (r1 < r0);
                carry = c0 | c1;
                c[i] = r1;
        }
        if (carry || (dv_cmp(c, shared_prime, RLC_FP_DIGS) != RLC_LT)) {
                carry = fp_subn_low(c, c, shared_prime);
        }
//        printf("returning from fp_addm_low...\n");
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_add(fp_t c, const fp_t a, const fp_t b) {
        fp_addm_low(c, a, b);
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t fp_prime_get_mod8(){
 return 3;
}
__device__
#if INLINE == 0
__noinline__
#endif
int fp_prime_get_qnr(){
 return -1;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_dblm_low(dig_t *c, const dig_t *a) {
        int i;
        dig_t carry, c0, c1, r0, r1;

        carry = 0;
        for (i = 0; i < RLC_FP_DIGS; i++, a++) {
                r0 = (*a) + (*a);
                c0 = (r0 < (*a));
                r1 = r0 + carry;
                c1 = (r1 < r0);
                carry = c0 | c1;
                c[i] = r1;
        }
        if (carry || (dv_cmp(c, shared_prime, RLC_FP_DIGS) != RLC_LT)) {
                carry = fp_subn_low(c, c, shared_prime);
        }
}       

__device__
#if INLINE == 0
__noinline__
#endif
void fp_dbl(fp_t c, const fp_t a) {
        fp_dblm_low(c, a);
}

__device__
#if INLINE == 0
__noinline__
#endif
void fp2_sqr_basic(fp2_t c, fp2_t a) {
 fp_t t0, t1, t2;

// printf("fp2_sqr_basic a: %" PRIu64 "\n", *a[0]);
// printf("fp2_sqr_basic a: %" PRIu64 "\n", *a[1]);

 t0 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
 t1 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
 t2 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));

 /* t0 = (a_0 + a_1). */
 fp_add(t0, a[0], a[1]);
 /* t1 = (a_0 - a_1). */
 fp_sub(t1, a[0], a[1]);
 /* t1 = a_0 + u^2 * a_1. */
 for (int i = -1; i > fp_prime_get_qnr(); i--) {
  fp_sub(t1, t1, a[1]);
 }
 for (int i = 1; i < fp_prime_get_qnr(); i++) {
  fp_add(t1, t1, a[1]);
 }
 if (fp_prime_get_qnr() == -1) {
  /* t2 = 2 * a_0. */
  fp_dbl(t2, a[0]);
  /* c_1 = 2 * a_0 * a_1. */
  fp_mul(c[1], t2, a[1]);
  /* c_0 = a_0^2 + a_1^2 * u^2. */
  fp_mul(c[0], t0, t1);
 } else {
 /* c_1 = a_0 * a_1. */
  fp_mul(c[1], a[0], a[1]);
 /* c_0 = a_0^2 + a_1^2 * u^2. */
  fp_mul(c[0], t0, t1);
 for (int i = -1; i > fp_prime_get_qnr(); i--) {
  fp_add(c[0], c[0], c[1]);
 }
 for (int i = 1; i < fp_prime_get_qnr(); i++) {
  fp_add(c[0], c[0], c[1]);
 }
 /* c_1 = 2 * a_0 * a_1. */
  fp_dbl(c[1], c[1]);
 }
 /* c = c_0 + c_1 * u. */
 free(t0);
 free(t1);
 free(t2);
// printf("fp2_sqr_basic  RESULTS c: %" PRIu64 "\n", *c[0]);
// printf("fp2_sqr_basic  RESULTS c: %" PRIu64 "\n", *c[1]);

}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t fp_addd_low(dig_t *c, const dig_t *a, const dig_t *b) {
        int i;
        dig_t carry, c0, c1, r0, r1;

        carry = 0;
        for (i = 0; i < 2 * RLC_FP_DIGS; i++, a++, b++) {
                r0 = (*a) + (*b);
                c0 = (r0 < (*a));
                r1 = r0 + carry;
                c1 = (r1 < r0);
                carry = c0 | c1;
                c[i] = r1;
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_addc_low(dig_t *c, const dig_t *a, const dig_t *b) {
        dig_t carry = fp_addd_low(c, a, b);

        if (carry || (dv_cmp(c + RLC_FP_DIGS, shared_prime, RLC_FP_DIGS) != RLC_LT)) {
                carry = fp_subn_low(c + RLC_FP_DIGS, c + RLC_FP_DIGS, shared_prime);
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_subc_low(dig_t *c, const dig_t *a, const dig_t *b) {
        int i;
        dig_t carry, r0, diff;

        /* Zero the carry. */
        carry = 0;
        for (i = 0; i < 2 * RLC_FP_DIGS; i++, a++, b++) {
                diff = (*a) - (*b);
                r0 = diff - carry;
                carry = ((*a) < (*b)) || (carry && !diff);
                c[i] = r0;
        }
        if (carry) {
                fp_addn_low(c + RLC_FP_DIGS, c + RLC_FP_DIGS, shared_prime);
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_rdc(fp_t c, dv_t a) {
        fp_rdc_basic(c, a);
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_read_raw(bn_t a, const dig_t *raw, int len) {
 bn_grow(a, len); 
 a->used = len;  
 a->sign = RLC_POS;
 dv_copy(a->dp, raw, len);
 bn_trim(a);
} 
//__device__
//#if INLINE == 0
//__noinline__
//#endif
//void util_print(const char *format, ...) {
//        va_list list;
//        va_start(list, format);
//        vprintf(format, list);
//        fflush(stdout);
//        va_end(list);
//}
__device__
#if INLINE == 0
__noinline__
#endif
void util_print_dig(dig_t a, int pad) {
if (pad) {
 printf("%.16" PRIX64, (uint64_t) a);
 } else {
 printf("%" PRIX64, (uint64_t) a);
}
///if (pad) {
/// util_print("%.16" PRIX64, (uint64_t) a);
/// } else {
/// util_print("%" PRIX64, (uint64_t) a);
///}
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_print(const fp_t a) {
        int i;
        bn_t t;


 t  = (bn_t ) malloc(sizeof(bn_st));
 t->dp = (dig_t* ) malloc(RLC_FP_DIGS * sizeof(dig_t));
 t->alloc = RLC_FP_DIGS;
 t->sign = RLC_POS;

//#if FP_RDC == MONTY
//                if (a != fp_prime_get()) {
//                        fp_prime_back(t, a);
//                } else {
//                        bn_read_raw(t, a, RLC_FP_DIGS);
//                }
//#else
                bn_read_raw(t, a, RLC_FP_DIGS);
//#endif

                for (i = RLC_FP_DIGS - 1; i > 0; i--) {
                        if (i >= t->used) {
                                util_print_dig(0, 1);
                        } else {
                                util_print_dig(t->dp[i], 1);
                        }
                        printf(" ");
                }
                util_print_dig(t->dp[0], 1);
                printf("\n");

// Ez lehet hogy majd okoz memóriaszivárgást...
//                bn_free(t);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp2_print(fp2_t a) {
        fp_print(a[0]);
        fp_print(a[1]);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp2_mul_basic(fp2_t c, fp2_t a, fp2_t b) {
	dv_t t0, t1, t2, t3, t4;

        t0 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
        t1 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
        t2 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
        t3 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
        t4 = (dv_t ) malloc( (RLC_DV_DIGS + RLC_PAD(RLC_DV_BYTES)/(RLC_DIG / 8))*sizeof(dig_t));
//        printf("now in fp2_mul_basic... \n");
//        fp2_print(c);
//        fp2_print(a);
//        fp2_print(b);
////        printf ("a0 %" PRIu64 "\n", *a[0]);
////        printf ("a1 %" PRIu64 "\n", *a[1]);
////        printf ("b0 %" PRIu64 "\n", *b[0]);
////        printf ("b1 %" PRIu64 "\n", *b[1]);
        /* Karatsuba algorithm. */
        /* t2 = a_0 + a_1, t1 = b_0 + b_1. */


        fp_add(t2, a[0], a[1]);
//        printf("t2:");
//        fp_print(t2);
        fp_add(t1, b[0], b[1]);
//        printf("b0:");
//        fp_print(b[0]);
//        printf("b1:");
//        fp_print(b[1]);
//        printf("t1:");
//        fp_print(t1);
////        printf ("t2 %" PRIu64 "\n", *t2);
////        printf ("t1 %" PRIu64 "\n", *t1);

        /* t3 = (a_0 + a_1) * (b_0 + b_1). */
        fp_muln_low(t3, t2, t1);
////        printf("t3:");
//        fp_print(t3);

////        printf ("t3 %" PRIu64 "\n", *t3);

        /* t0 = a_0 * b_0, t4 = a_1 * b_1. */
        fp_muln_low(t0, a[0], b[0]);
//        printf("t0:");
//        fp_print(t0);
////        printf ("t0 %" PRIu64 "\n", *t0);

        fp_muln_low(t4, a[1], b[1]);
//        printf("t4:");
//        fp_print(t4);
////        printf ("t4 %" PRIu64 "\n", *t4);


        /* t2 = (a_0 * b_0) + (a_1 * b_1). */
        fp_addc_low(t2, t0, t4);
//        printf("t2:");
//        fp_print(t2);

////        printf ("t2 %" PRIu64 "\n", *t2);

        /* t1 = (a_0 * b_0) + i^2 * (a_1 * b_1). */
        fp_subc_low(t1, t0, t4);
//        printf("t1:");
//        fp_print(t1);

////        printf ("t1 %" PRIu64 "\n", *t1);

        /* t1 = u^2 * (a_1 * b_1). */
        for (int i = -1; i > fp_prime_get_qnr(); i--) {
         fp_subc_low(t1, t1, t4);
        }
        for (int i = 1; i < fp_prime_get_qnr(); i++) {
         fp_addc_low(t1, t1, t4);
        }
        /* c_0 = t1 mod p. */
//        printf("t1:");
//        fp_print(t1);
        fp_rdc(c[0], t1);

// TODO debug fp_rdc valszeg ebben van a hiba!

        /* t4 = t3 - t2. */
	fp_subc_low(t4, t3, t2);
//        printf("t4:");
//        fp_print(t4);

////        printf ("t4 %" PRIu64 "\n", *t4);
	/* c_1 = t4 mod p. */
	fp_rdc(c[1], t4);

//       printf("result in fp2_mul_basic... \n");
//       printf ("%" PRIu64 "\n", *c[0]);
//       printf ("%" PRIu64 "\n", *c[1]);
//        fp2_print(c);
 free(t0);
 free(t1);
 free(t2);
 free(t3);
 free(t4);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_exp(fp_t c, const fp_t a, const bn_t b) {
 fp_exp_basic(c,a,b);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_inv(fp_t c, const fp_t a) {
        fp_inv_basic(c, a);
//        fp_inv_exgcd(c, a);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp2_add_basic(fp2_t c, fp2_t a, fp2_t b) {
  fp_add_basic(c[0], a[0], b[0]);
  fp_add_basic(c[1], a[1], b[1]);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_neg_basic(fp_t c, const fp_t a) {
        if (fp_is_zero(a)) {
                fp_zero(c);
        } else {
                fp_subn_low(c, shared_prime, a);
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_neg(fp_t c, const fp_t a) {
  fp_neg_basic(c,a);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_mul_dig(fp_t c, const fp_t a, dig_t b) {
        dv_t t;
        fp_prime_conv_dig(t, b);
        fp_mul(c, a, t);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp2_neg(fp2_t c, fp2_t a) {
        fp_neg(c[0], a[0]);
        fp_neg(c[1], a[1]);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp2_inv(fp2_t c, fp2_t a) {
	fp_t t0, t1;

//       printf("INPUT in fp2_inv... \n");
//       printf ("a0 %" PRIu64 "\n", *(a[0]) );
//       printf ("a1 %" PRIu64 "\n", *(a[0] + 1) );
//       printf ("a2 %" PRIu64 "\n", *(a[0] + 2) );
//       printf ("a3 %" PRIu64 "\n", *(a[0] + 3) );
//       printf ("a4 %" PRIu64 "\n", *(a[0] + 4) );
//       printf ("a5 %" PRIu64 "\n", *(a[0] + 5) );

// printf("1. fp2_inv ...\n");
        t0 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
// printf("2. fp2_inv ...\n");
        t1 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
// printf("3. fp2_inv ...\n");
		/* t0 = a_0^2, t1 = a_1^2. */
		fp_sqr(t0, a[0]);
//       printf("1. t0 ... \n");
//       printf ("t00 %" PRIu64 "\n", *(t0) );
//       printf ("t01 %" PRIu64 "\n", *(t0 + 1) );
//       printf ("t02 %" PRIu64 "\n", *(t0 + 2) );
//       printf ("t03 %" PRIu64 "\n", *(t0 + 3) );
//       printf ("t04 %" PRIu64 "\n", *(t0 + 4) );
//       printf ("t05 %" PRIu64 "\n", *(t0 + 5) );
// printf("4. fp2_inv ...\n");
		fp_sqr(t1, a[1]);
//       printf("2. t1 ... \n");
//       printf ("t10 %" PRIu64 "\n", *(t1) );
//       printf ("t11 %" PRIu64 "\n", *(t1 + 1) );
//       printf ("t12 %" PRIu64 "\n", *(t1 + 2) );
//       printf ("t13 %" PRIu64 "\n", *(t1 + 3) );
//       printf ("t14 %" PRIu64 "\n", *(t1 + 4) );
//       printf ("t15 %" PRIu64 "\n", *(t1 + 5) );

// printf("5. fp2_inv ...\n");
		/* t1 = 1/(a_0^2 + a_1^2). */
#ifndef FP_QNRES
		if (fp_prime_get_qnr() != -1) {
			if (fp_prime_get_qnr() == -2) {
// printf("6. fp2_inv ...\n");
				fp_dbl(t1, t1);
//       printf("3. t1 ... \n");
//       printf ("t10 %" PRIu64 "\n", *(t1) );
//       printf ("t11 %" PRIu64 "\n", *(t1 + 1) );
//       printf ("t12 %" PRIu64 "\n", *(t1 + 2) );
//       printf ("t13 %" PRIu64 "\n", *(t1 + 3) );
//       printf ("t14 %" PRIu64 "\n", *(t1 + 4) );
//       printf ("t15 %" PRIu64 "\n", *(t1 + 5) );

// printf("7. fp2_inv ...\n");
				fp_add(t0, t0, t1);
//       printf("4. t0 ... \n");
//       printf ("t00 %" PRIu64 "\n", *(t0) );
//       printf ("t01 %" PRIu64 "\n", *(t0 + 1) );
//       printf ("t02 %" PRIu64 "\n", *(t0 + 2) );
//       printf ("t03 %" PRIu64 "\n", *(t0 + 3) );
//       printf ("t04 %" PRIu64 "\n", *(t0 + 4) );
//       printf ("t05 %" PRIu64 "\n", *(t0 + 5) );

			} else {
				if (fp_prime_get_qnr() < 0) {
// printf("8. fp2_inv ...\n");
					fp_mul_dig(t1, t1, -fp_prime_get_qnr());
//       printf("5. t1 ... \n");
//       printf ("t10 %" PRIu64 "\n", *(t1) );
//       printf ("t11 %" PRIu64 "\n", *(t1 + 1) );
//       printf ("t12 %" PRIu64 "\n", *(t1 + 2) );
//       printf ("t13 %" PRIu64 "\n", *(t1 + 3) );
//       printf ("t14 %" PRIu64 "\n", *(t1 + 4) );
//       printf ("t15 %" PRIu64 "\n", *(t1 + 5) );

// printf("9. fp2_inv ...\n");
					fp_add(t0, t0, t1);
//       printf("6. t0 ... \n");
//       printf ("t00 %" PRIu64 "\n", *(t0) );
//       printf ("t01 %" PRIu64 "\n", *(t0 + 1) );
//       printf ("t02 %" PRIu64 "\n", *(t0 + 2) );
//       printf ("t03 %" PRIu64 "\n", *(t0 + 3) );
//       printf ("t04 %" PRIu64 "\n", *(t0 + 4) );
//       printf ("t05 %" PRIu64 "\n", *(t0 + 5) );

				} else {
// printf("10. fp2_inv ...\n");
					fp_mul_dig(t1, t1, fp_prime_get_qnr());
//       printf("8. t1 ... \n");
//       printf ("t10 %" PRIu64 "\n", *(t1) );
//       printf ("t11 %" PRIu64 "\n", *(t1 + 1) );
//       printf ("t12 %" PRIu64 "\n", *(t1 + 2) );
//       printf ("t13 %" PRIu64 "\n", *(t1 + 3) );
//       printf ("t14 %" PRIu64 "\n", *(t1 + 4) );
//       printf ("t15 %" PRIu64 "\n", *(t1 + 5) );

					fp_sub(t0, t0, t1);
//       printf("7. t0 ... \n");
//       printf ("t00 %" PRIu64 "\n", *(t0) );
//       printf ("t01 %" PRIu64 "\n", *(t0 + 1) );
//       printf ("t02 %" PRIu64 "\n", *(t0 + 2) );
//       printf ("t03 %" PRIu64 "\n", *(t0 + 3) );
//       printf ("t04 %" PRIu64 "\n", *(t0 + 4) );
//       printf ("t05 %" PRIu64 "\n", *(t0 + 5) );
				}
			}
		} else {

			fp_add(t0, t0, t1);
//       printf("10. t0 ... \n");
//       printf ("t00 %" PRIu64 "\n", *(t0) );
//       printf ("t01 %" PRIu64 "\n", *(t0 + 1) );
//       printf ("t02 %" PRIu64 "\n", *(t0 + 2) );
//       printf ("t03 %" PRIu64 "\n", *(t0 + 3) );
//       printf ("t04 %" PRIu64 "\n", *(t0 + 4) );
//       printf ("t05 %" PRIu64 "\n", *(t0 + 5) );

		}
#else
		fp_add(t0, t0, t1);
//       printf("11. t0 ... \n");
//       printf ("t00 %" PRIu64 "\n", *(t0) );
//       printf ("t01 %" PRIu64 "\n", *(t0 + 1) );
//       printf ("t02 %" PRIu64 "\n", *(t0 + 2) );
//       printf ("t03 %" PRIu64 "\n", *(t0 + 3) );
//       printf ("t04 %" PRIu64 "\n", *(t0 + 4) );
//       printf ("t05 %" PRIu64 "\n", *(t0 + 5) );
#endif
// printf("11. fp2_inv ...\n");
		fp_inv(t1, t0);
//       printf("9. t1 ... \n");
//       printf ("t10 %" PRIu64 "\n", *(t1) );
//       printf ("t11 %" PRIu64 "\n", *(t1 + 1) );
//       printf ("t12 %" PRIu64 "\n", *(t1 + 2) );
//       printf ("t13 %" PRIu64 "\n", *(t1 + 3) );
//       printf ("t14 %" PRIu64 "\n", *(t1 + 4) );
//       printf ("t15 %" PRIu64 "\n", *(t1 + 5) );

// printf("12. fp2_inv ...\n");
		/* c_0 = a_0/(a_0^2 + a_1^2). */
		fp_mul(c[0], a[0], t1);
// printf("13. fp2_inv ...\n");
		/* c_1 = - a_1/(a_0^2 + a_1^2). */
		fp_mul(c[1], a[1], t1);
// printf("14. fp2_inv ...\n");
		fp_neg(c[1], c[1]);
// printf("15. fp2_inv ...\n");
 free(t0);
 free(t1);
// printf("leaving fp2_inv ...\n");
//       printf("result in fp2_inv... \n");
//       printf ("c0 %" PRIu64 "\n", *(c[0]) );
//       printf ("c1 %" PRIu64 "\n", *(c[0] + 1) );
//       printf ("c2 %" PRIu64 "\n", *(c[0] + 2) );
//       printf ("c3 %" PRIu64 "\n", *(c[0] + 3) );
//       printf ("c4 %" PRIu64 "\n", *(c[0] + 4) );
//       printf ("c5 %" PRIu64 "\n", *(c[0] + 5) );

 
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t fp_add1_low(dig_t *c, const dig_t *a, dig_t digit) {
        int i;
        dig_t carry, r0;

        carry = digit;
        for (i = 0; i < RLC_FP_DIGS && carry; i++, a++, c++) {
                r0 = (*a) + carry;
                carry = (r0 < carry);
                (*c) = r0;
        }
        for (; i < RLC_FP_DIGS; i++, a++, c++) {
                (*c) = (*a);
        }
        return carry;
}

__device__
#if INLINE == 0
__noinline__
#endif
void dv_copy_cond(dig_t *c, const dig_t *a, int digits, dig_t cond) {
        dig_t mask, t;

        mask = -cond;
        for (int i = 0; i < digits; i++) {
                t = (a[i] ^ c[i]) & mask;
                c[i] ^= t;
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_add_dig(fp_t c, const fp_t a, dig_t b) {
#if FP_RDC == MONTY
        if (b == 1) {
// TODO is this OK?
//                fp_add(c, a, core_get()->one.dp);
                fp_add(c, a, shared_one);
        } else {
                fp_t t;

                fp_null(t); 

                        fp_new(t);

                        fp_set_dig(t, b);
                        fp_add(c, a, t);
        }
#else
        dig_t carry;

        carry = fp_add1_low(c, a, b);
        if (carry || dv_cmp(c, shared_prime, RLC_FP_DIGS) != RLC_LT) {
                carry = fp_subn_low(c, c, shared_prime);
        }
#endif
}

__device__
#if INLINE == 0
__noinline__
#endif
void fp2_add_dig(fp2_t c, const fp2_t a, dig_t dig) {
        fp_add_dig(c[0], a[0], dig);
        fp_copy(c[1], a[1]);
}
__device__
#if INLINE == 0
__noinline__
#endif
/**
 * Simplified SWU mapping.
 */
#define EP2_MAP_COPY_COND(O, I, C)                                                       \
        do {                                                                                 \
                dv_copy_cond(O[0], I[0], RLC_FP_DIGS, C);                                        \
                dv_copy_cond(O[1], I[1], RLC_FP_DIGS, C);                                        \
        } while (0)


__device__
#if INLINE == 0
__noinline__
#endif
/* caution: this function overwrites k, which it uses as an auxiliary variable */
int fp2_sgn0(const fp2_t t, bn_t k) {
        const int t_0_zero = fp_is_zero(t[0]);

//        printf("1. fp2_sgn0 called, now iside...\n");
        fp_prime_back(k, t[0]);
//        printf("2. fp2_sgn0 called, now iside...\n");
        const int t_0_neg = bn_get_bit(k, 0);
//        printf("3. fp2_sgn0 called, now iside...\n");

        fp_prime_back(k, t[1]);
//        printf("4. fp2_sgn0 called, now iside...\n");
        const int t_1_neg = bn_get_bit(k, 0);
//        printf("5. fp2_sgn0 called, now iside...\n");

        /* t[0] == 0 ? sgn0(t[1]) : sgn0(t[0]) */
        return t_0_neg | (t_0_zero & t_1_neg);
}

__device__
#if INLINE == 0
__noinline__
#endif
int fp2_is_zero(fp2_t a) {
        return fp_is_zero(a[0]) && fp_is_zero(a[1]);
}

__device__
#if INLINE == 0
__noinline__
#endif
void fp2_copy(fp2_t c, fp2_t a) {

//  if( c == NULL){
//    printf("c has problems.... \n");
//   }
//   else{
//    printf("c is OK .... \n");
//   } 
//
//  if( a == NULL){
//    printf("a has problems.... \n");
//   }
//   else{
//    printf("a is OK .... \n");
//   } 
//
//        printf("1. fp_copy call...\n");
        fp_copy(c[0], a[0]);
//        printf("2. fp_copy call...\n");
        fp_copy(c[1], a[1]);
}
__device__
#if INLINE == 0
__noinline__
#endif
int dv_cmp_const(const dig_t *a, const dig_t *b, int size) {
        dig_t r = 0;

        for (int i = 0; i < size; i++) {
                r |= a[i] ^ b[i];
        }

        return (r == 0 ? RLC_EQ : RLC_NE);
}
__device__
#if INLINE == 0
__noinline__
#endif
int fp_cmp(const fp_t a, const fp_t b) {
        return dv_cmp_const(a, b, RLC_FP_DIGS);
}
__device__
#if INLINE == 0
__noinline__
#endif
int fp_cmp_dig(const fp_t a, dig_t b) {
        fp_t t;
        int r = RLC_EQ;

        fp_null(t);
        fp_new(t);
        fp_prime_conv_dig(t, b);
        r = fp_cmp(a, t);

        return r;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_rsh(bn_t c, const bn_t a, int bits) {
        int digits = 0;

        bn_copy(c, a);

        if (bits <= 0) {
                return;
        }

        RLC_RIP(bits, digits, bits);

        if (digits > 0) {
                dv_rshd(c->dp, a->dp, a->used, digits);
        }
        c->used = a->used - digits;
        c->sign = a->sign;

        if (c->used > 0 && bits > 0) {
                if (digits == 0 && c != a) {
                        bn_rshb_low(c->dp, a->dp + digits, a->used - digits, bits);
                } else {
                        bn_rshb_low(c->dp, c->dp, c->used, bits);
                }
        }
        bn_trim(c);
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_add_dig(bn_t c, const bn_t a, dig_t b) {
 dig_t carry;
 bn_grow(c, a->used);
 if (a->sign == RLC_POS) {
  carry = bn_add1_low(c->dp, a->dp, b, a->used);
  if (carry) {
   bn_grow(c, a->used + 1);
   c->dp[a->used] = carry;
  }
  c->used = a->used + carry;
  c->sign = RLC_POS;
  } 
 else {
  /* If a < 0 && |a| >= b, compute c = -(|a| - b). */
  if (a->used > 1 || a->dp[0] >= b) {
   carry = bn_sub1_low(c->dp, a->dp, b, a->used);
   c->used = a->used;
   c->sign = RLC_NEG;
   } else {
   /* If a < 0 && |a| < b. */
   if (a->used == 1) {
    c->dp[0] = b - a->dp[0];
    } else {
     c->dp[0] = b;
    }
    c->used = 1;
    c->sign = RLC_POS;
    }
 }
 bn_trim(c);
}
__device__
#if INLINE == 0
__noinline__
#endif
int bn_is_even(const bn_t a) {
        if (bn_is_zero(a)) {
                return 1;
        }
        if ((a->dp[0] & 0x01) == 0) {
                return 1;
        }
        return 0;
}  
__device__
#if INLINE == 0
__noinline__
#endif
int fp_srt(fp_t c, const fp_t a) {

	bn_t e;

	fp_t t0;
	fp_t t1;

	int r = 0;

        e  = (bn_t ) malloc(sizeof(bn_st));
        e->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
        e->alloc = RLC_BN_SIZE;
        e->sign = RLC_POS;

        t0 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
        t1 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));

	bn_null(e);
	fp_null(t0);
	fp_null(t1);

	if (fp_is_zero(a)) {
		fp_zero(c);
		return 1;
	}

		bn_new(e);
		fp_new(t0);
		fp_new(t1);

		/* Make e = p. */
		e->used = RLC_FP_DIGS;
		dv_copy(e->dp, shared_prime, RLC_FP_DIGS);

//		if (fp_prime_get_mod8() == 3 || fp_prime_get_mod8() == 7) {
			/* Easy case, compute a^((p + 1)/4). */
			bn_add_dig(e, e, 1);
			bn_rsh(e, e, 2);

			fp_exp(t0, a, e);
			fp_sqr(t1, t0);
			r = (fp_cmp(t1, a) == RLC_EQ);
			fp_copy(c, t0);
//		} else {
//			int f = 0, m = 0;
//
//			/* First, check if there is a root. Compute t1 = a^((p - 1)/2). */
//			bn_rsh(e, e, 1);
//			fp_exp(t0, a, e);
//
//			if (fp_cmp_dig(t0, 1) != RLC_EQ) {
//				/* Nope, there is no square root. */
//				r = 0;
//			} else {
//				r = 1;
//				/* Find a quadratic non-residue modulo p, that is a number t2
//				 * such that (t2 | p) = t2^((p - 1)/2)!= 1. */
//				do {
//// TODO is this deterministic?
//					fp_rand(t1);
//					fp_exp(t0, t1, e);
//				} while (fp_cmp_dig(t0, 1) == RLC_EQ);
//
//				/* Write p - 1 as (e * 2^f), odd e. */
//				bn_lsh(e, e, 1);
//				while (bn_is_even(e)) {
//					bn_rsh(e, e, 1);
//					f++;
//				}
//
//				/* Compute t2 = t2^e. */
//				fp_exp(t1, t1, e);
//
//				/* Compute t1 = a^e, c = a^((e + 1)/2) = a^(e/2 + 1), odd e. */
//				bn_rsh(e, e, 1);
//				fp_exp(t0, a, e);
//				fp_mul(e->dp, t0, a);
//				fp_sqr(t0, t0);
//				fp_mul(t0, t0, a);
//				fp_copy(c, e->dp);
//
//				while (1) {
//					if (fp_cmp_dig(t0, 1) == RLC_EQ) {
//						break;
//					}
//					fp_copy(e->dp, t0);
//					for (m = 0; (m < f) && (fp_cmp_dig(t0, 1) != RLC_EQ); m++) {
//						fp_sqr(t0, t0);
//					}
//					fp_copy(t0, e->dp);
//					for (int i = 0; i < f - m - 1; i++) {
//						fp_sqr(t1, t1);
//					}
//					fp_mul(c, c, t1);
//					fp_sqr(t1, t1);
//					fp_mul(t0, t0, t1);
//					f = m;
//				}
//			}
//		}
	return r;
}
__device__
#if INLINE == 0
__noinline__
#endif
dig_t fp_rsh1_low(dig_t *c, const dig_t *a) {
        int i;
        dig_t r, carry;

        c += RLC_FP_DIGS - 1;
        a += RLC_FP_DIGS - 1;
        carry = 0;
        for (i = RLC_FP_DIGS - 1; i >= 0; i--, a--, c--) {
                /* Get the least significant bit. */
                r = *a & 0x01;
                /* Shift the operand and insert the carry. */
                carry <<= RLC_DIG - 1;
                *c = (*a >> 1) | carry;
                /* Update the carry. */
                carry = r;
        }
        return carry;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_hlv_basic(fp_t c, const fp_t a) {
        dig_t carry = 0;

        if (a[0] & 1) {
                carry = fp_addn_low(c, a, shared_prime);
        } else {
                fp_copy(c, a);
        }
        fp_rsh1_low(c, c);
        if (carry) {
                c[RLC_FP_DIGS - 1] ^= ((dig_t)1 << (RLC_DIG - 1));
        }
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp_hlv(fp_t c, const fp_t a) {
 fp_hlv_basic(c,a);
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp2_zero(fp2_t a) {
        fp_zero(a[0]);
        fp_zero(a[1]);
}

__device__
#if INLINE == 0
__noinline__
#endif
int fp2_srt(fp2_t c, fp2_t a) {
 int r = 0;
 fp_t t0;        
 fp_t t1;        
 fp_t t2;

 t0 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
 t1 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
 t2 = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));

// printf("1. fp2_srt \n");
 if (fp2_is_zero(a)) {
  fp2_zero(c);
  free(t0);
  free(t1);
  free(t2);
  return 1;
 }

 if (fp_is_zero(a[1])) {
  /* special case: either a[0] is square and sqrt is purely 'real'
  * or a[0] is non-square and sqrt is purely 'imaginary' */
  r = 1;
  if (fp_srt(t0, a[0])) {
   fp_copy(c[0], t0);
   fp_zero(c[1]);
//   printf("2. fp2_srt \n");
  } 
  else {
  /* Compute a[0]/i^2. */
#ifdef FP_QNRES
   fp_copy(t0, a[0]);
#else
   if (fp_prime_get_qnr() == -2) {
    fp_hlv(t0, a[0]);
   } 
   else {
//    printf("3. fp2_srt \n");
    fp_set_dig(t0, -fp_prime_get_qnr());
    fp_inv(t0, t0);
//    printf("4. fp2_srt \n");
    fp_mul(t0, t0, a[0]);
   }
#endif
  fp_neg(t0, t0);
//    printf("5. fp2_srt \n");
  fp_zero(c[0]);
//    printf("6. fp2_srt \n");
  if (!fp_srt(c[1], t0)) {
   /* should never happen! */
   printf("Problem in squaring field elements...\n");
  }
  }
 } 
 else {
  /* t0 = a[0]^2 - i^2 * a[1]^2 */
  fp_sqr(t0, a[0]);
  fp_sqr(t1, a[1]);
 
  for (int i = -1; i > fp_prime_get_qnr(); i--) {
   fp_add(t0, t0, t1);
  }
 
  fp_add(t0, t0, t1);
 
  if (fp_srt(t1, t0)) {
   /* t0 = (a_0 + sqrt(t0)) / 2 */
   fp_add(t0, a[0], t1);
   fp_hlv(t0, t0);
   if (!fp_srt(t2, t0)) {
    /* t0 = (a_0 - sqrt(t0)) / 2 */
    fp_sub(t0, a[0], t1);
    fp_hlv(t0, t0);
    if (!fp_srt(t2, t0)) {
     /* should never happen! */
     printf("Problem in squaring field elements...\n");
    }
    
    /* c_0 = sqrt(t0) */
    fp_copy(c[0], t2);
    /* c_1 = a_1 / (2 * sqrt(t0)) */
    fp_dbl(t2, t2);
    fp_inv(t2, t2);
    fp_mul(c[1], a[1], t2);
    r = 1;
   }
  }
  free(t0);
  free(t1);
  free(t2);
  return r;
 }
}
__device__
#if INLINE == 0
__noinline__
#endif
char util_conv_char(dig_t i) {
#if WSIZE == 8 || WSIZE == 16
        /* Avoid tables to save up some memory. This is not performance-critical. */
        if (i < 10) {
                return i + '0';
        }
        if (i < 36) {
                return (i - 10) + 'A';
        }
        if (i < 62) {
                return (i - 36) + 'a';
        }
        if (i == 62) {
                return '+';
        } else {
                return '/';
        }
#else
        /* Use a table. */
        static const char conv_table[] =
                        "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz+/";
        return conv_table[i];
#endif
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_mul_dig(bn_t c, const bn_t a, dig_t b) {
 bn_grow(c, a->used + 1);
 c->sign = a->sign;
 c->dp[a->used] = bn_mul1_low(c->dp, a->dp, b, a->used);
 c->used = a->used + 1;
 bn_trim(c);
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_read_str(bn_t a, const char *str, int len, int radix) {
        int sign, i, j;
        char c;

// printf("1. bn_read_str \n");
        bn_zero(a);
// printf("2. bn_read_str \n");

        if (radix < 2 || radix > 64) {
                printf("radix < 2 || radix > 64 in bn_read_str...");
                return;
        }

// printf("3. bn_read_str \n");
        j = 0;  
        if (str[0] == '-') {
                j++;
                sign = RLC_NEG;
        } else {
                sign = RLC_POS;
        }
//        printf("RLC_DIG %d \n", RLC_DIG);
//        printf("len %d \n", len);
//        printf("radix %d \n", radix);
//        printf("util_bits_dig(radix), %d \n", util_bits_dig(radix));
//        printf("len * util_bits_dig(radix), %d \n", len * util_bits_dig(radix));
//        printf("RLC_CEIL(len * util_bits_dig(radix), RLC_DIG) %d \n ", RLC_CEIL(len * util_bits_dig(radix), RLC_DIG) );

// printf("4. bn_read_str \n");
                bn_grow(a, RLC_CEIL(len * util_bits_dig(radix), RLC_DIG));
// printf("5. bn_read_str \n");
//        printf("j %d \n", j);
                while (j < len) {
                        if (str[j] == 0) {
                                break;
                        }
                        c = (char)((radix < 36) ? RLC_UPP(str[j]) : str[j]);
// printf("6. bn_read_str \n");
                        for (i = 0; i < 64; i++) {
// printf("7. bn_read_str \n");
                                if (c == util_conv_char(i)) {
                                        break;
                                }
                        }

                        if (i < radix) {
// printf("8. bn_read_str \n");
//                                printf("a->used %d \n", a->used);
                                bn_mul_dig(a, a, (dig_t)radix);
// printf("9. bn_read_str \n");
                                bn_add_dig(a, a, (dig_t)i);
                        } else {
                                break;
                        }
                        j++;
                }

                a->sign = sign;
// printf("10. bn_read_str \n");
}

__device__ 
#if INLINE == 0
__noinline__
#endif
void bn_print(const bn_t a) {
        int i;

        if (a->sign == RLC_NEG) {
               printf("-");
        }
        if (a->used == 0) {
                printf("0\n");
        } else {
                util_print_dig(a->dp[a->used - 1], 0);
                for (i = a->used - 2; i >= 0; i--) {
                        util_print_dig(a->dp[i], 1);
                }
                printf("\n");
        }
}
__device__ 
#if INLINE == 0
__noinline__
#endif
void fp_read_str(fp_t a, const char *str, int len, int radix) {
 bn_t t;

 t  = (bn_t ) malloc(sizeof(bn_st));
 t->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
 t->alloc = RLC_BN_SIZE;
 t->sign = RLC_POS;

 bn_read_str(t, str, len, radix);
//                printf("Printing t in fp_read_str..\n");
//                bn_print(t);

 if (bn_is_zero(t)) {
  fp_zero(a);
 } 
 else {
  if (t->used == 1) {
//                printf("Printing a 1 in fp_read_str..\n");
//                fp_print(a);
   fp_prime_conv_dig(a, t->dp[0]);
//                printf("Printing a 2 in fp_read_str..\n");
//                fp_print(a);

   if (bn_sign(t) == RLC_NEG) {
    fp_neg(a, a);
//                printf("Printing a 3 in fp_read_str..\n");
//                fp_print(a);

   }
  } 
  else {
   fp_prime_conv(a, t);
//                printf("Printing a 4 in fp_read_str..\n");
//                fp_print(a);

   }
 }
 free(t->dp);
 free(t);
}
__device__
#if INLINE == 0
__noinline__
#endif
size_t strlen_cuda(const char *str)
{
	const char *s;
	for (s = str; *s; ++s)
		;
	return (s - str);
}
__device__
#if INLINE == 0
__noinline__
#endif
char * strchr_cuda(register const char *s, int c)
{
  do {
    if (*s == c)
      {
        return (char*)s;
      }
  } while (*s++);
  return (0);
}
/**
 * Reads a sequence of polynomial coefficients from semicolon separated string.
 *
 * @param[out] coeffs		- the resulting coefficients.
 * @param[in] str			- the input string.
 */
__device__ __noinline__
__noinline__
int ep2_curve_get_coeffs(fp2_t *coeffs, const char *str) {
	int degree = 0;
	unsigned offset = 0;

	if (str[0] == '\0') {
		/* need nonzero strlen */
		printf("str[0] == '\0' in ep2_curve_get_coeffs... \n");
		return 0;
	}
	for (; degree < RLC_EPX_CTMAP_MAX; ++degree) {
		/* first coeff */
		const char *end = strchr_cuda(str + offset, ',');
		if (end == NULL) {
			/* should not happen --- means there's no second coeff */
			printf("end == NULL...\n");
		}
		unsigned len = end - str - offset;
		fp_read_str(coeffs[degree][0], str + offset, len, 16);
		offset += len + 1; /* move to after ',' */

		/* second coeff */
		end = strchr_cuda(str + offset, ';');
		if (end == NULL) {
			/* last one */
			fp_read_str(coeffs[degree][1], str + offset, strlen_cuda(str + offset), 16);
			break;
		}
		len = end - str - offset;
		fp_read_str(coeffs[degree][1], str + offset, len, 16);
		offset += len + 1; /* move to after ';' */
	}
	if (degree == RLC_EPX_CTMAP_MAX) {
		/* ran out of space before converting all coeffs */
		printf("degree == RLC_EPX_CTMAP_MAX...\n");
	}
	return degree;
}
__device__
#if INLINE == 0
__noinline__
#endif
void fp2_set_dig(fp2_t a, dig_t b) {
        fp_set_dig(a[0], b);
        fp_zero(a[1]);
}
/**
 * Normalizes a point represented in projective coordinates.
 *
 * @param r                     - the result.
 * @param p                     - the point to normalize.
 */
__device__
#if INLINE == 0
__noinline__
#endif
static void ep2_norm_imp(ep2_t r, ep2_t p, int inverted) {
 if (p->coord != BASIC) {
  fp2_t t0, t1;

 t0[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
 t0[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

 t1[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
 t1[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

  if (inverted) {
   fp2_copy(t1, p->z);
  } else {
   fp2_inv(t1, p->z);
  }
   fp2_sqr_basic(t0, t1);
   fp2_mul_basic(r->x, p->x, t0);
   fp2_mul_basic(t0, t0, t1);
   fp2_mul_basic(r->y, p->y, t0);
   fp2_set_dig(r->z, 1);
  }
  r->coord = BASIC;
}
__device__
#if INLINE == 0
__noinline__
#endif
int ep2_is_infty(ep2_t p) {
        return (fp2_is_zero(p->z) == 1);
}
__device__
#if INLINE == 0
__noinline__
#endif
void ep2_set_infty(ep2_t p) {
        fp2_zero(p->x);
        fp2_zero(p->y);
        fp2_zero(p->z);
        p->coord = BASIC;
}
__device__
#if INLINE == 0
__noinline__
#endif
void ep2_copy(ep2_t r, ep2_t p) {
        fp2_copy(r->x, p->x);
        fp2_copy(r->y, p->y);
        fp2_copy(r->z, p->z);
        r->coord = p->coord;
}
__device__
#if INLINE == 0
__noinline__
#endif
void ep2_norm(ep2_t r, ep2_t p) {
 if (ep2_is_infty(p)) {
  ep2_set_infty(r);
 return;
 }
 if (p->coord == BASIC) {
  /* If the point is represented in affine coordinates, we just copy it. */
  ep2_copy(r, p);
 }
#if EP_ADD == PROJC || !defined(STRIP)
 ep2_norm_imp(r, p, 0);
#endif
}
// Evaluate a polynomial using Horner's rule
__device__
#if INLINE == 0
__noinline__
#endif
static void fp2_eval(fp2_t c, fp2_t a, fp2_t *coeffs, int deg) {
 fp2_copy(c, coeffs[deg]);                    
 for (int i = deg; i > 0; --i) {            
  fp2_mul_basic(c, c, a);                  
  fp2_add_basic(c, c, coeffs[i - 1]);     
  }                                       
}
__device__
#if INLINE == 0
__noinline__
#endif
void isogeny_map(ep2_t p){
 fp2_t t0, t1, t2, t3;

 t0[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
 t0[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

 t1[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
 t1[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

 t2[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
 t2[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

 t3[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
 t3[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

 /* XXX need to add real support for input projective points */
 printf("Normalizing the coordinates...\n");
 if (p->coord != BASIC) {            
  ep2_norm(p, p);                     
 }                                     
 iso2_t coeffs = shared_coeffs;
 printf("Evaluating the polynomial...\n");
 /* numerators */                             
 fp2_eval(t0, p->x, coeffs->xn, coeffs->deg_xn);
 fp2_eval(t1, p->x, coeffs->yn, coeffs->deg_yn);
 /* denominators */                           
 fp2_eval(t2, p->x, coeffs->yd, coeffs->deg_yd);
 fp2_eval(t3, p->x, coeffs->xd, coeffs->deg_xd);
 /* normalize if necessary */  

/* Y = Ny * Dx * Z^2. */                                                                                        
 fp2_mul_basic(p->y, p->y, t1);
 fp2_mul_basic(p->y, p->y, t3);
 /* Z = Dx * Dy, t1 = Z^2. */                                                                            \
 fp2_mul_basic(p->z, t2, t3);
 fp2_sqr_basic(t1, p->z);
 fp2_mul_basic(p->y, p->y, t1);
 /* X = Nx * Dy * Z. */                                                                                          \
 fp2_mul_basic(p->x, t0, t2);
 fp2_mul_basic(p->x, p->x, p->z);
 p->coord = PROJC;

}

__device__ __noinline__
void map_scalar_to_curve(ep2_t p, fp2_t t){
 fp2_t t0, t1, t2, t3;

 char str[4 * RLC_FP_BYTES + 1];
 fp_t *mBoverA;
 fp_t *a;
 fp_t *b;
 fp_t *u;  

 a = (fp_t*)malloc(RLC_BN_SIZE * sizeof(dig_t));
 b = (fp_t*)malloc(RLC_BN_SIZE * sizeof(dig_t));
 u = (fp_t*)malloc(RLC_BN_SIZE * sizeof(dig_t));
 
 t0[0] = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
 t0[1] = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));

 t1[0] = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
 t1[1] = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));

 t2[0] = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
 t2[1] = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));

 t3[0] = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));
 t3[1] = (fp_t)malloc(RLC_BN_SIZE * sizeof(dig_t));

 mBoverA = shared_c[0];
 a       = shared_c[2];
 b       = shared_c[3];

// Precomputes constants
// ez a beolvasás valszeg nem jó...
 shared_map_u[0] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
 shared_map_u[1] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));

 memcpy(str, B12_P381_MAPU0, sizeof(B12_P381_MAPU0));
 fp_read_str(shared_map_u[0], str, 2, 16);
 memcpy(str, B12_P381_MAPU1, sizeof(B12_P381_MAPU1));
 fp_read_str(shared_map_u[1], str, 2, 16);

 u       = shared_map_u;

// print_line();
// printf("u after fp_read_str ... \n");
// print_multiple_precision(shared_map_u[0],1);
// print_multiple_precision(shared_map_u[1],1);
// fp2_print(shared_map_u);
 printf("now precomputing the isomap constants...\n");
 /* SSWU map constants */
 /* constants 3 and 4 are a and b for the curve or isogeny */
 shared_c[0][0]   = (fp_t )malloc(RLC_BN_SIZE * sizeof(dig_t));
 shared_c[0][1]   = (fp_t )malloc(RLC_BN_SIZE * sizeof(dig_t));
 shared_c[1][0]   = (fp_t )malloc(RLC_BN_SIZE * sizeof(dig_t));
 shared_c[1][1]   = (fp_t )malloc(RLC_BN_SIZE * sizeof(dig_t));
 shared_c[2][0]   = (fp_t )malloc(RLC_BN_SIZE * sizeof(dig_t));
 shared_c[2][1]   = (fp_t )malloc(RLC_BN_SIZE * sizeof(dig_t));
 shared_c[3][0]   = (fp_t )malloc(RLC_BN_SIZE * sizeof(dig_t));
 shared_c[3][1]   = (fp_t )malloc(RLC_BN_SIZE * sizeof(dig_t));
 fp2_copy(shared_c[2], shared_coeffs->a);
 fp2_copy(shared_c[3], shared_coeffs->b);
 /* constant 1: -b / a */
 fp2_neg(shared_c[0], shared_c[2]);     /* c1 = -a */
 fp2_inv(shared_c[0], shared_c[0]);     /* c1 = -1 / a */
 fp2_mul_basic(shared_c[0], shared_c[0], shared_c[3]); /* c1 = -b / a */
// Precomputation ends here //
// Compute the SSWU Map
 printf("Computing the SSWU map ...\n");

 printf("t: \n");
 fp2_print(t);
 fp2_sqr_basic(t0, t);
 printf("t^2: \n");
 fp2_print(t0);

 printf("u:  \n");
 fp2_print(u);
 fp2_mul_basic(t0, t0, u);  /* t0 = u * t^2 */
 printf("u * t^2: \n");
 fp2_print(t0);
 fp2_sqr_basic(t1, t0);     /* t1 = u^2 * t^4 */
 printf("u^2 * t^4: \n");
 fp2_print(t1);
 fp2_add_basic(t2, t1, t0); /* t2 = u^2 * t^4 + u * t^2 */ 
 printf("u^2 * t^4 + u * t^2: \n");
 fp2_print(t2);
 printf("Computing the SSWU map finished...\n");

 /* handle the exceptional cases */  
 /* XXX(rsw) should be done projectively */   
 {                                            
  const int e1 = fp2_is_zero(t2);     
  fp2_neg(t3, u);         /* t3 = -u */      
  EP2_MAP_COPY_COND(t2, t3, e1);        /* exception: -u instead of u^2t^4 + ut^2 */ 
         fp2_inv(t2, t2);        /* t2 = -1/u or 1/(u^2 * t^4 + u*t^2) */
         fp2_add_dig(t3, t2, 1); /* t3 = 1 + t2 */                                
         EP2_MAP_COPY_COND(t2, t3, e1 == 0);      /* only add 1 if t2 != -1/u */         
 }                                                                              
 /* e1 goes out of scope */                                                    
                                                                              
 /* compute x1, g(x1) */                                                    
 printf("compute x1, g(x1)... \n");
 fp2_mul_basic(p->x, t2, mBoverA); /* -B / A * (1 + 1 / (u^2 * t^4 + u * t^2)) */
 fp2_sqr_basic(p->y, p->x);        /* x^2 */                                    
 fp2_add_basic(p->y, p->y, a);     /* x^2 + a */                               
 fp2_mul_basic(p->y, p->y, p->x);  /* x^3 + a x */                            
 fp2_add_basic(p->y, p->y, b);     /* x^3 + a x + b */                       
 printf("compute x1, g(x1) finished... \n");
 /* compute x2, g(x2) */                                            
 printf("compute x2, g(x2) ... \n");
 fp2_mul_basic(t2, t0, p->x); /* t2 = u * t^2 * x1 */                    
 fp2_mul_basic(t1, t0, t1);   /* t1 = u^3 * t^6 */                      
 fp2_mul_basic(t3, t1, p->y); /* t5 = g(t2) = u^3 * t^6 * g(p->x) */   
 printf("compute x2, g(x2) finished... \n");
//  /* XXX(rsw)                                                               */   
//  /* This should be done in constant time and without computing 2 sqrts.    */  
//  /* Avoiding a second sqrt relies on knowing the 2-adicity of the modulus. */ 
  if (!fp2_srt(p->y, p->y)) {                                                 
          /* try x2, g(x2) */                                                
          fp2_copy(p->x, t2);                                               
          if (!fp2_srt(p->y, t3)) {                                        
                  printf("+++++++++ Error +++++++++ \n");
                  printf("!fp2_srt(p->y, t3) in MAP calculation...\n");
                  printf("++++++++ !Error! ++++++++++ \n");
          }                                                              
  }    
  fp2_set_dig(p->z, 1);
  p->coord = BASIC;  
  free(t0[0]);
  free(t0[1]);
  free(t1[0]);
  free(t1[1]);
  free(t2[0]);
  free(t2[1]);
  free(t3[0]);
  free(t3[1]);
  return;
// }
}

/**
 * Configures a constant-time hash-to-curve function based on an isogeny map.
 *
 * @param[in] a0_str                    - the string representing the 1st element of the 'a' coefficient.
 * @param[in] a1_str                    - the string representing the 2nd element of the 'a' coefficient.
 * @param[in] b0_str                    - the string representing the 1st element of the 'b' coefficient.
 * @param[in] b1_str                    - the string representing the 2nd element of the 'b' coefficient.
 * @param[in] xn_str                    - the string representing the x numerator coefficients.
 * @param[in] xd_str                    - the string representing the x denominator coefficients.
 * @param[in] yn_str                    - the string representing the y numerator coefficients.
 * @param[in] yd_str                    - the string representing the y denominator coefficients.
 */
/* declaring this function inline suppresses unused function warnings */
__device__ __noinline__
void ep2_curve_set_ctmap(const char *a0_str, const char *a1_str, const char *b0_str, const char *b1_str, const char *xn_str, const char *xd_str, const char *yn_str, const char *yd_str) {

        shared_coeffs = (iso2_t )malloc(sizeof(iso2_st));

        shared_coeffs->a[0] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
        shared_coeffs->a[1] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));

        shared_coeffs->b[0] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
        shared_coeffs->b[1] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));

        for (unsigned i = 0; i < RLC_EPX_CTMAP_MAX; ++i) {
                shared_coeffs->xn[i][0] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
                shared_coeffs->xn[i][1] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
                shared_coeffs->xd[i][0] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
                shared_coeffs->xd[i][1] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
                shared_coeffs->yn[i][0] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
                shared_coeffs->yn[i][1] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
                shared_coeffs->yd[i][0] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
                shared_coeffs->yd[i][1] = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
        }


//        printf("a0_str: %d\n", strlen_cuda(a0_str));
//        printf("a1_str: %d\n", strlen_cuda(a1_str));
//        printf("b0_str: %d\n", strlen_cuda(b0_str));
//        printf("b1_str: %d\n", strlen_cuda(b1_str));
//
//        /* coefficients of isogenous curve */
//        fp_read_str(iso->a[0], a0_str, strlen_cuda(a0_str), 16);
//        fp_read_str(iso->a[1], a1_str, strlen_cuda(a1_str), 16);
//        fp_read_str(iso->b[0], b0_str, strlen_cuda(b0_str), 16);
//        fp_read_str(iso->b[1], b1_str, strlen_cuda(b1_str), 16);

        fp_read_str(shared_coeffs->a[0], a0_str, 1, 16);
        fp_read_str(shared_coeffs->a[1], a1_str, 2, 16);
        fp_read_str(shared_coeffs->b[0], b0_str, 3, 16);
        fp_read_str(shared_coeffs->b[1], b1_str, 3, 16);

        printf("Done reading a and b...\n");
        /* isogeny map coeffs */
        shared_coeffs->deg_xn = ep2_curve_get_coeffs(shared_coeffs->xn, xn_str);
        shared_coeffs->deg_xd = ep2_curve_get_coeffs(shared_coeffs->xd, xd_str);
        shared_coeffs->deg_yn = ep2_curve_get_coeffs(shared_coeffs->yn, yn_str);
        shared_coeffs->deg_yd = ep2_curve_get_coeffs(shared_coeffs->yd, yd_str);
        printf("Leaving ep2_curve_set_ctmap a and b...\n");
        return;

//        printf("xn: %d\n", shared_coeffs->deg_xn);
//        printf("xd: %d\n", shared_coeffs->deg_xd);
//        printf("yn: %d\n", shared_coeffs->deg_yn);
//        printf("yd: %d\n", shared_coeffs->deg_yd);
}




__device__
#if INLINE == 0
__noinline__
#endif
void signmessage(bn_t e, bn_t e2, int sequence){
 uint64_t carry;
 int neg;
 bn_t u, m, tt;
 fp_t r;
 fp2_t ttt;
 bn_st conv;
 bn_st one;
 ep2_t p;
 printf("sequence:  %d \n", sequence);
// print_line();
// printf("shared_prime: \n");
// print_multiple_precision(shared_prime,6);
// print_line();
 m = (bn_t )malloc(sizeof(bn_t)); 
 m->dp = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t)); 
 m->used = 6;
 for(int i=0; i < 6; i++){
  m->dp[i] = shared_prime[i];
 } 



 m->alloc = RLC_BN_SIZE;
 m->sign = RLC_POS;
 shared_prime_bn = m;
// Initialize u (the result of the Montgomery reduction) 
  u = (bn_t )malloc(sizeof(bn_t)); 
  u->dp = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t)); 
  u->used = 1;
  u->dp[0] = 0;
  u->alloc = RLC_BN_SIZE;
  u->sign = RLC_POS;
/////////////////////////////////////////////////////////////////////////
  one.dp = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
  one.used = 1;
  one.alloc = RLC_BN_SIZE;
  one.sign = RLC_POS;
  conv.dp = (dig_t *)malloc(RLC_BN_SIZE * sizeof(dig_t));
  conv.used = 1;
  conv.alloc = RLC_BN_SIZE;
  conv.sign = RLC_POS;
// Call the Montgomery reduction function

  bn_mod_pre_monty(u, m);

  shared_u = u->dp;
// Multiply by the Montgomery reduced prime (u)
  bn_set_dig(&one, 1);

// printf("\n---- one --- \n");
// for(int i=0; i < one.used; i++){
//  printf("one %d %" PRIu64 " \n",i, one.dp[i]);
// }
//  printf("1. one: \n");
//  print_multiple_precision(one.dp,6);
//  print_line();
  bn_lsh(&one, &one, RLC_FP_DIGS * RLC_DIG);
// printf("\n---- one 1--- \n");
// for(int i=0; i < one.used; i++){
//  printf("one %d %" PRIu64 " \n",i, one.dp[i]);
// }
//  printf("2. one: \n");
//  print_multiple_precision(one.dp,6);
//  print_line();

// Calculate 1 mod p
  bn_mod_basic(&one, &one, m);
// printf("\n---- one 2--- \n");
// for(int i=0; i < one.used; i++){
//  printf("one %d %" PRIu64 " \n",i, one.dp[i]);
// }
//  printf("3. one: \n");
//  print_multiple_precision(one.dp,6);
//  print_line();
  r = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
  fp_add_basic(r, one.dp, one.dp);
// printf("\n---- one 3--- \n");
// for(int i=0; i < one.used; i++){
//  printf("one %d %" PRIu64 " \n",i, one.dp[i]);
// }
  tt = (bn_t ) malloc(sizeof(bn_st));
  tt->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
  tt->alloc = RLC_BN_SIZE;
  tt->sign = RLC_POS;
  bn_set_dig(u, RLC_FP_DIGS);
  bn_lsh(u, u, RLC_DIG_LOG);
//  printf("r: \n");
//  print_multiple_precision(r,6);
//  print_line();
//
//  printf("u: \n");
//  print_multiple_precision(u->dp,6);
//  print_line();
// printf("\n---- u  --- \n");
// for(int i=0; i < u->used; i++){
//  printf("u %d %" PRIu64 " \n",i, u->dp[i]);
// }
// printf("\n---- t --- \n");

  fp_exp_basic(conv.dp, r, u);
  conv.used = RLC_FP_DIGS;
  bn_trim(&(conv));

// printf("\n---- conv --- \n");
// for(int i=0; i < conv.used; i++){
//  printf("conv %d %" PRIu64 " \n",i, conv.dp[i]);
// }
/////////////////////////////////////////////////////////////////////////
// Ez nagyon gany megoldas...
  shared_conv[0] = conv.dp[0];
  shared_conv[1] = conv.dp[1];
  shared_conv[2] = conv.dp[2];
  shared_conv[3] = conv.dp[3];
  shared_conv[4] = conv.dp[4];
  shared_conv[5] = conv.dp[5];
  shared_one[0] = one.dp[0];
  shared_one[1] = one.dp[1];
  shared_one[2] = one.dp[2];
  shared_one[3] = one.dp[3];
  shared_one[4] = one.dp[4];
  shared_one[5] = one.dp[5];
  ttt[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
  ttt[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

// e and e2 contain the sha256 converted message
// the message is reduced and loaded into ttt


//  printf("shared_conv: \n");
//  print_multiple_precision(shared_conv,6);
//  print_line();
//
//  printf("shared_one: \n");
//  print_multiple_precision(shared_one,6);
//  print_line();
//
//  printf("shared_u: \n");
//  print_multiple_precision(shared_u,6);
//  print_line();

  fp_prime_conv(ttt[0], e);
  fp_prime_conv(ttt[1], e2);

  printf("bID %d thID: %d ttt[0] %" PRIu64 " ttt[1] %" PRIu64 "\n",blockIdx.x, threadIdx.x,  *ttt[0], *ttt[1]);
  fp_print(ttt[0]);
  fp_print(ttt[1]);

/////////////////////////////////////////////////////////////////////////
  /* sign of t */                                                                
  neg = fp2_sgn0(ttt, e);
  printf("Setting the isogeny map ...\n");
// Calculate parameters for the curve isogeny
  ep2_curve_set_ctmap(B12_P381_ISO_A0, B12_P381_ISO_A1, B12_P381_ISO_B0, B12_P381_ISO_B1, B12_P381_ISO_XN, B12_P381_ISO_XD, B12_P381_ISO_YN, B12_P381_ISO_YD);
// Map scalar to B12_P381
  printf("Mapping the scalar to the curve ...\n");

  p = (ep2_t)malloc(sizeof(ep2_st));
  p->x[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
  p->x[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

  p->y[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
  p->y[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));

  p->z[0] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));
  p->z[1] = (fp_t)malloc((RLC_FP_DIGS + RLC_PAD(RLC_FP_BYTES)/(RLC_DIG / 8)) * sizeof(dig_t));



  map_scalar_to_curve(p, ttt);

  printf("Finished mapping the scalar to the curve ...\n");

  neg = neg != fp2_sgn0(p->y, e);
  fp2_neg(ttt, p->y);
  dv_copy_cond(p->y[0], ttt[0], RLC_FP_DIGS, neg);
  dv_copy_cond(p->y[1], ttt[1], RLC_FP_DIGS, neg);

// Now apply the isogeny map
//  print_line();
//  printf("The point before applying the isogeny map... \n");
//
//  printf("x coordinate: \n");
//  printf("p->x[0] %" PRIu64 "\n", *(p->x[0] ));
//  printf("p->x[1] %" PRIu64 "\n", *(p->x[0] + 1));
//  printf("p->x[2] %" PRIu64 "\n", *(p->x[0] + 2));
//  printf("p->x[3] %" PRIu64 "\n", *(p->x[0] + 3));
//  printf("p->x[4] %" PRIu64 "\n", *(p->x[0] + 4));
//  printf("p->x[5] %" PRIu64 "\n", *(p->x[0] + 5));
//  printf("y coordinate: \n");
//  printf("p->y[0] %" PRIu64 "\n", *(p->y[0] ));
//  printf("p->y[1] %" PRIu64 "\n", *(p->y[0] + 1));
//  printf("p->y[2] %" PRIu64 "\n", *(p->y[0] + 2));
//  printf("p->y[3] %" PRIu64 "\n", *(p->y[0] + 3));
//  printf("p->y[4] %" PRIu64 "\n", *(p->y[0] + 4));
//  printf("p->y[5] %" PRIu64 "\n", *(p->y[0] + 5));
//  printf("z coordinate: \n");
//  printf("p->z[0] %" PRIu64 "\n", *(p->z[0] ));
//  printf("p->z[1] %" PRIu64 "\n", *(p->z[0] + 1));
//  printf("p->z[2] %" PRIu64 "\n", *(p->z[0] + 2));
//  printf("p->z[3] %" PRIu64 "\n", *(p->z[0] + 3));
//  printf("p->z[4] %" PRIu64 "\n", *(p->z[0] + 4));
//  printf("p->z[5] %" PRIu64 "\n", *(p->z[0] + 5));

  printf("Now applying the isogeny map... \n");

  isogeny_map(p);

  uint64_t px0;
  uint64_t px1;
  uint64_t px2;
  uint64_t px3;
  uint64_t px4;
  uint64_t px5;

  uint64_t py0;
  uint64_t py1;
  uint64_t py2;
  uint64_t py3;
  uint64_t py4;
  uint64_t py5;

  uint64_t pz0;
  uint64_t pz1;
  uint64_t pz2;
  uint64_t pz3;
  uint64_t pz4;
  uint64_t pz5;

  if( sequence == 2){
////////////////////////////////////////
  px0 = 12876831369499511095;
  px1 = 669865624959273240;
  px2 = 5295314185012811767;
  px3 = 10535942465924952507;
  px4 = 6749632078625001325;
  px5 = 544070190095032775;

  py0 = 13807866252442817510;
  py1 = 1206778827755322241;
  py2 = 6227792413039013907;
  py3 = 12409653748090456027;
  py4 = 15923766313813229816;
  py5 = 1280940689572399341;

  pz0 = 14186990084672407827;
  pz1 = 17414337241577503165;
  pz2 = 17737221398210816263;
  pz3 = 5096200064592250870;
  pz4 = 6989538818909868085;
  pz5 = 1219742961965905715;
  }

  if( sequence == 1){
//////////////////////////////////////////
   px0 =  16603095906398444624;
   px1 =  14847369377878545419;
   px2 =  8937774108357643346;
   px3 =  17574689352006813814;
   px4 =  4620856641434282496;
   px5 =  1793872408053411627;
   
   py0 =  17684262036518551128;
   py1 =  18386928879118654222;
   py2 =  13389195836363238096;
   py3 =  6284561072484266232;
   py4 =  5663343310858039123;
   py5 =  1864692639812907776;
   
   pz0 =  10838604725761032207;
   pz1 =  4762092640527260469;
   pz2 =  14549356420920161935;
   pz3 =  5090114536724173654;
   pz4 =  8300404609496668738;
   pz5 =  1618339427510023074;

  }

  if( sequence == 3){
//////////////////////////////////////////
  px0 = 4522175503006271926;
  px1 = 7628860187923640081;
  px2 = 15714645553301507193;
  px3 = 3513625221767549538;
  px4 = 9416525119631435704;
  px5 = 942297724353119737;

  py0 = 7064085492358342813;
  py1 = 3011720167030279849;
  py2 = 6475112925619391838;
  py3 = 17963467974497784593;
  py4 = 9265567426871135047;
  py5 = 1266056669242111252;

  pz0 = 7727439254583007199;
  pz1 = 15108752542132982959;
  pz2 = 12915093228206905201;
  pz3 = 12683266112564391531;
  pz4 = 16970245155421669696;
  pz5 = 585935835826096346;
  }

  printf("Isogeny map applied successfully... \n");
////////////////////////////////////////////////////////////////////
  printf("The resulting point (P) on BLS12-381: \n");
  printf("x coordinate: \n");
  printf("p->x[0] %" PRIu64 "  %" PRIu64 "\n", *(p->x[0]), px0 );
  printf("p->x[1] %" PRIu64 "  %" PRIu64 "\n", *(p->x[0] + 1), px1);
  printf("p->x[2] %" PRIu64 "  %" PRIu64 "\n", *(p->x[0] + 2), px2);
  printf("p->x[3] %" PRIu64 "  %" PRIu64 "\n", *(p->x[0] + 3), px3);
  printf("p->x[4] %" PRIu64 "  %" PRIu64 "\n", *(p->x[0] + 4), px4);
  printf("p->x[5] %" PRIu64 "  %" PRIu64 "\n", *(p->x[0] + 5), px5);
  printf("y coordinate: \n");  
  printf("p->y[0] %" PRIu64 "  %" PRIu64 "\n", *(p->y[0] ), py0);
  printf("p->y[1] %" PRIu64 "  %" PRIu64 "\n", *(p->y[0] + 1), py1);
  printf("p->y[2] %" PRIu64 "  %" PRIu64 "\n", *(p->y[0] + 2), py2);
  printf("p->y[3] %" PRIu64 "  %" PRIu64 "\n", *(p->y[0] + 3), py3);
  printf("p->y[4] %" PRIu64 "  %" PRIu64 "\n", *(p->y[0] + 4), py4);
  printf("p->y[5] %" PRIu64 "  %" PRIu64 "\n", *(p->y[0] + 5), py5);
  printf("z coordinate: \n");  
  printf("p->z[0] %" PRIu64 "  %" PRIu64 "\n", *(p->z[0] ), pz0);
  printf("p->z[1] %" PRIu64 "  %" PRIu64 "\n", *(p->z[0] + 1), pz1);
  printf("p->z[2] %" PRIu64 "  %" PRIu64 "\n", *(p->z[0] + 2), pz2);
  printf("p->z[3] %" PRIu64 "  %" PRIu64 "\n", *(p->z[0] + 3), pz3);
  printf("p->z[4] %" PRIu64 "  %" PRIu64 "\n", *(p->z[0] + 4), pz4);
  printf("p->z[5] %" PRIu64 "  %" PRIu64 "\n", *(p->z[0] + 5), pz5);

  /* compare sign of y to sign of t; fix if necessary */
  printf("Deallocating memory ...\n");

  free(tt->dp);
  free(u->dp);
  free(m->dp);

  free(m);

  free(one.dp);
  free(conv.dp);
  free(r);
  free(tt);
  free(p->x[0]);
  free(p->x[1]);
  free(p->y[0]);
  free(p->y[1]);
  free(p->z[0]);
  free(p->z[1]);
  free(p);
  free(u);
  free(ttt[0]);
  free(ttt[1]);

  return;
}
__device__
#if INLINE == 0
__noinline__
#endif
void bn_read_bin(bn_t a, const uint8_t *bin, int len) {
 int i, j; 
 dig_t d = (RLC_DIG / 8); 
 int digs = (len % d == 0 ? len / d : len / d + 1);
 bn_grow(a, digs);
 bn_zero(a);
 a->used = digs;
 printf("\n-------------------\n");
 for (i = 0; i < len; i++) {
  printf("%02x",bin[i]);
 }
 printf("\n-------------------\n");

 for (i = 0; i < digs - 1; i++) {
         d = 0; 
         for (j = (RLC_DIG / 8) - 1; j >= 0; j--) {
                 d = d << 8;
                 d |= bin[len - 1 - (i * (RLC_DIG / 8) + j)];

         }
         a->dp[i] = d;
// printf("a->dp[i] %" PRIu64 " \n", a->dp[i]);


 } 
 d = 0; 
 for (j = (RLC_DIG / 8) - 1; j >= 0; j--) { 
         if ((int)(i * (RLC_DIG / 8) + j) < len) {
                 d = d << 8;
                 d |= bin[len - 1 - (i * (RLC_DIG / 8) + j)];
         }
 }
 a->dp[i] = d;
// printf("a->dp[i] %" PRIu64 " \n", a->dp[i]);
 a->sign = RLC_POS;
 bn_trim(a);


}

__device__
#if INLINE == 0
__noinline__
#endif
void convert_from_hexa(uint8_t *source_array, uint64_t *target_array, int max_index, int block_size){

  for(int index=0; index < max_index; index++){
   int target_array_index = max_index - index - 1;

   target_array[target_array_index] = 0;
   target_array[target_array_index] |= source_array[index*block_size+0];

//   printf("%" PRIu64 "\n", target_array[target_array_index]);

   target_array[target_array_index] = target_array[target_array_index] << 8;
//   printf("%" PRIu64 "\n", target_array[target_array_index]);
   target_array[target_array_index] |= source_array[index*block_size+1];
//   printf("%" PRIu64 "\n", target_array[target_array_index]);

   target_array[target_array_index] = target_array[target_array_index] << 8;
//   printf("%" PRIu64 "\n", target_array[target_array_index]);
   target_array[target_array_index] |= source_array[index*block_size+2];
//   printf("%" PRIu64 "\n", target_array[target_array_index]);

   target_array[target_array_index] = target_array[target_array_index] << 8;
//   printf("%" PRIu64 "\n", target_array[target_array_index]);
   target_array[target_array_index] |= source_array[index*block_size+3];
//   printf("%" PRIu64 "\n", target_array[target_array_index]);

   target_array[target_array_index] = target_array[target_array_index] << 8;
//   printf("%" PRIu64 "\n", target_array[target_array_index]);
   target_array[target_array_index] |= source_array[index*block_size+4];
//   printf("%" PRIu64 "\n", target_array[target_array_index]);

   target_array[target_array_index] = target_array[target_array_index] << 8;
//   printf("%" PRIu64 "\n", target_array[target_array_index]);
   target_array[target_array_index] |= source_array[index*block_size+5];
//   printf("%" PRIu64 "\n", target_array[target_array_index]);

   target_array[target_array_index] = target_array[target_array_index] << 8;
//   printf("%" PRIu64 "\n", target_array[target_array_index]);
   target_array[target_array_index] |= source_array[index*block_size+6];
//   printf("%" PRIu64 "\n", target_array[target_array_index]);

   target_array[target_array_index] = target_array[target_array_index] << 8;
//   printf("%" PRIu64 "\n", target_array[target_array_index]);
   target_array[target_array_index] |= source_array[index*block_size+7];
//   printf("%" PRIu64 "\n", target_array[target_array_index]);
  }

}
__global__
void saxpy(uint8_t *prime, uint64_t *prime2)
{
 bn_t e, e2;
   uint8_t  idx0;
   uint8_t  idx1;

    // mapping of ASCII characters to hex values
   const uint8_t hashmap[] =
   {
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, //  !"#$%&'
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ()*+,-./
     0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, // 01234567
     0x08, 0x09, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // 89:;<=>?
     0x00, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f, 0x00, // @ABCDEFG
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // HIJKLMNO
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // PQRSTUVW
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // XYZ[\]^_
     0x00, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f, 0x00, // `abcdefg
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // hijklmno
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // pqrstuvw
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // xyz{|}~.
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, // ........
     0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00  // ........
   };

 uint8_t *msg;

 char message_string_1[513] = "b629a33baa2f71304e6a1f84eed5ab383a23bb055b1442795bcd7ac4fba664c5e178dba9737570dd6ded5f73fd3fbbac25f559e84f2154d8ab0d32442da10a60fc830f54bbfc0b19ba723b0bc4177b96c5fc6aa77cee05ef80163fca2b5b92145c7004beef09abab3d52c6989da26ee0e8c4d63587b8e6127279d5abf4b520becfcd98c2163f82d7f1777d5559fc77ad040bbb8b933780211a5ef359f70788f95773612f69638cae550aed382d68a4be0c194139e7b3069126b2dad1d6e5d8fe5cfc8d5a90c783f1ebab25d095776172a66d9afdb16d7c289ad89c93dda54c7b0cf86991a200ff0e858573796cf396e6ae77470b4dd2d61267f5716de16b199f";
// 15CC3F292D66704A 62687D6E2DCB5913 7C9E20D357539F96 9F51EABD020D64E0 81792D01F15CC248 1D44777B8BAFC9FD ttt[0]
// 147765C676F3B800 6798BA4300F29F76 27CBE052A3D0397E CE0A4A7079E5EFEF 45DECCC08A4147E2 345BA7EC94B37852 ttt[1]

 char message_string_2[513] = "975a7a3edf0c907a8670af92ed36b3a1e94940ed8d4fe0e54592e0a4d6527b5bd6fd4cb9968d760b68be1dc82f576a6a73cc0714e02e353ad6d510f5dfc7f02479abf7ee20e927345cc36b408d3fcd05729fdf18f74f4ad91cc4bd50d3795fe5cfbbfb060689552b39e996fcece89e258b7db611a41c271216af110d493e81e96f9b1aa1696ef41c6573563e84c547de86f18d3ea897956dda7ca5101a47138c906602acf2ebd4cc1c8411b1e4f83825eaacbe54c9ed8a5ae2df3dd04bc77f223e03d78e10ca95d59de0bc047dd33e5a170473d8f70d94bf467ed9684a1ed05cff88779990ba1aa0832005af2a19be3cdd46e68094ed0ba34789c80f24d5f07f";
 char message_string_3[513] = "3d762157e3c4566456bb1a25654b4c17dcc15079d6343a54b76723a2da8580e22fcab914a229f2885d46ce3ac0beb3d1a64a26b1b166acb26b284b25586e1f8d0f3ee175ab69ad80ab1fb623623d1cd750b28c5ba6062d0573ab2b66a83457afce074f5179b8b849fc82d8957121c7bc73b48a64c59e3bd51533769bcb48a61190acf98407ea195ca53ec47b1261227f2fb2652436c094990482889d569b310991c4ae7dcddb375c956a705841a9c5fc87acef7c35f461b4f26d5031b3ce6857f90c78ce931c006f61a3410fef514e1070d07d15cc429d42a86edae22a3650777e94810e873728cc769704660d07a488d3d8efc503fb8c7fdc5de06743fb4936";
 char message_string_4[513] = "b629a33baa2f71304e6a1f84eed5ab383a23bb055b1442795bcd7ac4fba664c5e178dba9737570dd6ded5f73fd3fbbac25f559e84f2154d8ab0d32442da10a60fc830f54bbfc0b19ba723b0bc4177b96c5fc6aa77cee05ef80163fca2b5b92145c7004beef09abab3d52c6989da26ee0e8c4d63587b8e6127279d5abf4b520becfcd98c2163f82d7f1777d5559fc77ad040bbb8b933780211a5ef359f70788f95773612f69638cae550aed382d68a4be0c194139e7b3069126b2dad1d6e5d8fe5cfc8d5a90c783f1ebab25d095776172a66d9afdb16d7c289ad89c93dda54c7b0cf86991a200ff0e858573796cf396e6ae77470b4dd2d61267f5716de16b199f";
 convert_from_hexa(prime,prime2, 6, 8);
 for(int i=0; i < 6; i++){
  shared_prime[i] = prime2[i];
 } 
 msg = (uint8_t*)malloc(256*sizeof(uint8_t));

  e = (bn_t ) malloc(sizeof(bn_st));
  e->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
  e->alloc = RLC_BN_SIZE;
  e->sign = RLC_POS;

  e2 = (bn_t ) malloc(sizeof(bn_st));
  e2->dp = (dig_t* ) malloc(RLC_BN_SIZE * sizeof(dig_t));
  e2->alloc = RLC_BN_SIZE;
  e2->sign = RLC_POS;

////////////////////////////////////////////////////////////
 for(int i = 512; i >= 2; i -= 2){
  int j = 512 - i;
  idx0 = (uint8_t)message_string_1[j];
  idx1 = (uint8_t)message_string_1[j+1];
  int  k = i / 2 - 1;
  msg[k] = (uint8_t)(hashmap[idx0] << 4) | hashmap[idx1];
 }
 for(int i = 0; i<256; i++){
  printf("%02x",msg[i]);
 }

  printf("\n");

  bn_read_bin(e, msg, 64);
  bn_read_bin(e2, msg+64, 64);


  signmessage(e,e2, 1);
  return;


////////////////////////////////////////////////////////////
  for(int i = 512; i >= 2; i -= 2){
   int j = 512 - i;
   idx0 = (uint8_t)message_string_2[j];
   idx1 = (uint8_t)message_string_2[j+1];
   int  k = i / 2 - 1;
   msg[k] = (uint8_t)(hashmap[idx0] << 4) | hashmap[idx1];
  }

 for(int i = 0; i<256; i++){
  printf("%02x",msg[i]);
 }

  printf("\n");
  bn_read_bin(e, msg, 64);
  bn_read_bin(e2, msg+64, 64);

  signmessage(e,e2, 2);
////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////
  for(int i = 512; i >= 2; i -= 2){
   int j = 512 - i;
   idx0 = (uint8_t)message_string_3[j];
   idx1 = (uint8_t)message_string_3[j+1];
   int  k = i / 2 - 1;
   msg[k] = (uint8_t)(hashmap[idx0] << 4) | hashmap[idx1];
  }
 for(int i = 0; i<256; i++){
  printf("%02x",msg[i]);
 }

  printf("\n");

  bn_read_bin(e, msg, 64);
  bn_read_bin(e2, msg+64, 64);

  signmessage(e,e2, 3);
////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////
  for(int i = 512; i >= 2; i -= 2){
   int j = 512 - i;
   idx0 = (uint8_t)message_string_4[j];
   idx1 = (uint8_t)message_string_4[j+1];
   int  k = i / 2 - 1;
   msg[k] = (uint8_t)(hashmap[idx0] << 4) | hashmap[idx1];
  }
 for(int i = 0; i<256; i++){
  printf("%02x",msg[i]);
 }

  printf("\n");

  bn_read_bin(e, msg, 64);
  bn_read_bin(e2, msg+64, 64);

  signmessage(e,e2, 4);
////////////////////////////////////////////////////////////

  free(msg);
  free(e->dp);
  free(e2->dp);

  free(e);
  free(e2);
}

int main(void)
{
  uint8_t *msg, *d_msg;
  uint64_t *msg2, *d_msg2;
  uint8_t *msg_first, *d_msg_first;

  uint8_t *prime, *cuda_prime;
  uint64_t *prime_2, *cuda_prime_2;

  uint64_t *quotient, *remainder;

  msg = (uint8_t*)malloc(64*sizeof(uint8_t));
  hipMalloc(&d_msg, 64*sizeof(uint8_t)); 

  msg_first = (uint8_t*)malloc(64*sizeof(uint8_t));
  hipMalloc(&d_msg_first, 64*sizeof(uint8_t)); 

  msg2 = (uint64_t*)malloc((8+1)*sizeof(uint64_t));
  hipMalloc(&d_msg2, (8+1)*sizeof(uint64_t)); 

  prime = (uint8_t*)malloc(48*sizeof(uint8_t));
  hipMalloc(&cuda_prime, 48*sizeof(uint8_t)); 

  prime_2 = (uint64_t*)malloc((6 + 1)*sizeof(uint64_t));
  hipMalloc(&cuda_prime_2, (6 + 1)*sizeof(uint64_t)); 

  hipMalloc(&quotient, (6+1)*sizeof(uint64_t)); 
  hipMalloc(&remainder, (6+1)*sizeof(uint64_t)); 

  prime[47] = 0xAB;
  prime[46] = 0xAA;
  prime[45] = 0xFF;
  prime[44] = 0xFF;
  prime[43] = 0xFF;
  prime[42] = 0xFF;
  prime[41] = 0xFE;
  prime[40] = 0xB9;

  prime[39] = 0xFF;
  prime[38] = 0xFF;
  prime[37] = 0x53;
  prime[36] = 0xB1;
  prime[35] = 0xFE;
  prime[34] = 0xFF;
  prime[33] = 0xAB;
  prime[32] = 0x1E;

  prime[31] = 0x24;
  prime[30] = 0xF6;
  prime[29] = 0xB0;
  prime[28] = 0xF6;
  prime[27] = 0xA0;
  prime[26] = 0xD2;
  prime[25] = 0x30;
  prime[24] = 0x67;

  prime[23] = 0xBF;
  prime[22] = 0x12;
  prime[21] = 0x85;
  prime[20] = 0xF3;
  prime[19] = 0x84;
  prime[18] = 0x4B;
  prime[17] = 0x77;
  prime[16] = 0x64;

  prime[15] = 0xD7;
  prime[14] = 0xAC;
  prime[13] = 0x4B;
  prime[12] = 0x43;
  prime[11] = 0xB6;
  prime[10] = 0xA7;
  prime[9] = 0x1B;
  prime[8] = 0x4B;
  prime[7] = 0x9A;
  prime[6] = 0xE6;
  prime[5] = 0x7F;
  prime[4] = 0x39;
  prime[3] = 0xEA;
  prime[2] = 0x11;
  prime[1] = 0x01;
  prime[0] = 0x1A;

  hipMemcpy(d_msg, msg, 64*sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMemcpy(d_msg_first, msg_first, 64*sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMemcpy(cuda_prime, prime, 48*sizeof(uint8_t), hipMemcpyHostToDevice);

  size_t deviceLimit;
  gpuErrChk(hipDeviceGetLimit(&deviceLimit, hipLimitStackSize));
//  printf("Original Device stack size: %d\n", (int) deviceLimit);
    
  hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
  gpuErrChk(hipDeviceSetLimit(hipLimitStackSize, 1024));
  gpuErrChk(hipDeviceGetLimit(&deviceLimit, hipLimitStackSize));

  hipEvent_t start, stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventRecord(start,0);

  saxpy<<<NBLOCKS, NTHREADS>>>(cuda_prime, cuda_prime_2);


  hipEventCreate(&stop);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start,stop);
  printf("Elapsed time : %f ms\n" ,elapsedTime);

  hipMemcpy(msg2, d_msg2, 8*sizeof(uint64_t), hipMemcpyDeviceToHost);
  hipMemcpy(prime_2, cuda_prime_2, 6*sizeof(uint64_t), hipMemcpyDeviceToHost);
// h = SHA-256(msg) --> 8 uint64_t conversion done

  hipFree(cuda_prime);
  hipFree(d_msg);
  hipFree(d_msg2);
  free(prime);
  free(msg);
  free(msg2);
  hipDeviceReset();
}
